#include "hip/hip_runtime.h"
﻿#include "CudaBorder.cuh"

CudaBorder::CudaBorder()
{
}
CudaBorder::~CudaBorder()
{
	SaveDelete(PointType);
	SaveDelete(PointType_1D);
}

//////////////////////////////////////////////////////////////////////////
// GPU
//////////////////////////////////////////////////////////////////////////
__global__ static void NormalizaDataGPU(float* DataArray, float maxValue, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size)
		return;

	DataArray[id] /= maxValue;
}
__global__ static void findMaxAndMinPeak(float* DataArray, uchar* PointType, int rows, int cols, float MinGapPeakThreshold,  float MaxPeakThreshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows * cols)				// 超出範圍
		return;

	// width 判斷 1 ~ (width - 1)
	int colID = id % cols;
	if (1 >= colID || colID == (cols - 1))
		return;

	// 接著要去比周圍
	// 峰值判斷 (要比兩邊高，且峰值要高於某一個值，且左 或右差值，只有一端能高於這個值)
	float DiffLeft = DataArray[id] - DataArray[id - 1];
	float DiffRight = DataArray[id] - DataArray[id + 1];
	if (DiffLeft > 0 && DiffRight > 0
		&& DataArray[id] > MaxPeakThreshold)
		PointType[id] = 1;
	else if (DiffLeft < 0 && DiffRight < 0
		&& ((-DiffLeft > MinGapPeakThreshold) || (-DiffRight > MinGapPeakThreshold)))
		PointType[id] = 2;
}
__global__ static void ParseMaxMinPeak(uchar* PointType, int rows, int cols, int startIndex)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows)						// 超出範圍
		return;

	// 然後這邊要去 Skip 所有的 Min
	int lastMinID = -1;
	bool FindMax = false;

	// 為了要抓出 最大(有效)的 區間
	int Useful_Start = -1;
	int Useful_End = -1;
	int Useful_PeakCount = -1, tempPeakCount = 0;

	// 刪除多餘 min Peak
	for (int i = 0; i < startIndex; i++)
	{
		int currentID = id * cols + i;
		PointType[currentID] = 0;
	}
	for (int i = startIndex; i < cols; i++)
	{
		int currentID = id * cols + i;
		if (lastMinID == -1)			// 判斷是不適剛開始 or 找到 Max
		{
			// 要先去抓出第一個 Min
			if (PointType[currentID] == 2)
				lastMinID = i;
			else if (PointType[currentID] == 1)
				PointType[currentID] = 0;				// 這邊代表沒有遇到峰值，應該是雜訊了
		}
		else
		{
			// 已經抓到 min 了之後，要去濾掉其他的 min
			if (PointType[currentID] == 1)
			{
				// 抓到 Max
				FindMax = true;
				tempPeakCount++;
			}
			else if (FindMax && PointType[currentID] == 2)
			{
				// 抓到 Max 之後，又找到一個 Min
				if (Useful_PeakCount < tempPeakCount)
				{
					Useful_PeakCount = tempPeakCount;
					Useful_Start = lastMinID;
					Useful_End = i;
				}
				FindMax = false;
				tempPeakCount = 0;
				lastMinID = -1;
			}
			else if (!FindMax && PointType[currentID] == 2)
			{
				// 沒抓到 Max 只抓到 Min
				PointType[id * cols + lastMinID] = 0;
				lastMinID = i;
			}
		}
	}

	// 跑到最後結束，要再去判斷最後一個是否是多餘的 Min
	if (lastMinID != -1)
		PointType[id * cols + lastMinID] = 0;
}
__global__ static void TransforToImage(float* DataArray, uchar* OutArray, int rows, int cols)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows * cols)				// 判斷是否超出大小
		return;

	float data = ((DataArray[id] / (float)3.509173f) - (float)(3.39f / 3.509173f)) * 255;
	if (data >= 255)
		OutArray[id] = 255;
	else if (data <= 0)
		OutArray[id] = 0;
	else
		OutArray[id] = (unsigned char)data;
}
__global__ static void PickBestChoiceToArray(float* DataArray, uchar* PointType, int* PointType_1D, int rows, int cols, float Threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows)									// 判斷是否超出大小
		return;

	bool IsFindMin = false;							// 是否找到底端
	bool IsFindBorder = false;						// 是否找到邊界 (找到底端之後，要開始找邊界)
	float MinData;
	int offsetIndex = id * cols;
	//printf("%d\n", id);
	for (int i = 0; i < cols; i++)
	{
		if (!IsFindMin && PointType[i + offsetIndex] == 2)
		{
			IsFindMin = true;
			MinData = DataArray[i + offsetIndex];
		}
		else if (IsFindMin && DataArray[i + offsetIndex] - MinData > Threshold)
		{
			IsFindBorder = true;
			PointType_1D[id] = i;
			break;
		}
	}

	// 接這著要判斷是否找到邊界
	// 如果沒有找到邊界，就回傳 -1
	if (!IsFindBorder)
		PointType_1D[id] = -1;

}
__global__ static void StablizePointType(int * PointType_1D, int* OutputType_1D, int rows, int chooseSize)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows)									// 判斷是否超出大小
		return;

	// 由於雜點會有很多 Noise 要去雜訊用的
	int range = chooseSize / 2;
	int count = 0;
	int TotalHeight = 0;
	for (int i = -range; i <= range; i++)
	{
		if (0 > i + id || id + i >= rows)			// 超出邊界
			continue;
		else if (PointType_1D[id + i] == -1)		// 沒有資訊
			continue;
		else if (abs(PointType_1D[id + i] - PointType_1D[id]) > 30)		// 差距太大
			continue;
		else
		{
			TotalHeight += PointType_1D[id + i];
			count++;
		}
	}

	if (count >= range - 1)
		OutputType_1D[id] = TotalHeight / count;
	else
		OutputType_1D[id] = -1;
}

//////////////////////////////////////////////////////////////////////////
// CPU
//////////////////////////////////////////////////////////////////////////
void CudaBorder::Init(int rows, int cols)
{
	// 給值
	this->rows = rows;
	this->cols = cols;

	// 初始化
	SaveDelete(PointType);
	PointType = new uchar[rows * cols];
	SaveDelete(PointType_1D);
	PointType_1D = new int[rows];
}
void CudaBorder::GetBorderFromCuda(float** DataArray)
{
	#pragma region 前置判斷
	// 要先初始化
	assert(PointType_1D != NULL && PointType != NULL && rows != 0 && cols != 0);
	#pragma endregion
	#pragma region 開始時間
	clock_t time;
	time = clock();
	#pragma endregion
	#pragma region GPU Init
	float *GPU_DataArray;
	hipMalloc(&GPU_DataArray, sizeof(float) * rows * cols);
	hipMemcpy(GPU_DataArray, &DataArray[0][0], sizeof(float) * rows * cols, hipMemcpyHostToDevice);

	// 點的型別
	uchar* GPU_PointType;
	hipMalloc(&GPU_PointType, sizeof(uchar) * rows * cols);
	hipMemset(GPU_PointType, 0, sizeof(uchar) * rows * cols);
	#pragma endregion
	#pragma region 抓取最大值 每個除以最大值
	float maxValue = GetMaxValue(GPU_DataArray,  rows * cols);
	NormalizeData(GPU_DataArray, maxValue, rows * cols);

	// 找最大最小值
	findMaxAndMinPeak << < NumBlocks, NumThreads >> > (GPU_DataArray, GPU_PointType, rows, cols, MinGapPeakThreshold, MaxPeakThreshold);
	CheckCudaError();

	// Parse 一些連續最小值
	ParseMaxMinPeak << < NumBlocks_small, NumThreads_small >> > (GPU_PointType, rows, cols, StartIndex);
	CheckCudaError();

	// 抓出一維陣列
	int *GPU_PointType_1D, *GPU_tempPointType_1D;
	hipMalloc(&GPU_PointType_1D, sizeof(int) * rows);
	hipMalloc(&GPU_tempPointType_1D, sizeof(int) * rows);
	PickBestChoiceToArray << <NumBlocks_small, NumThreads_small >> > (GPU_DataArray, GPU_PointType, GPU_tempPointType_1D, rows, cols, 0.03f);
	CheckCudaError();

	StablizePointType << <NumBlocks_small, NumThreads_small >> > (GPU_tempPointType_1D, GPU_PointType_1D, rows, StablizeSize);
	CheckCudaError();

	// 把資料傳回 CPU
	hipMemcpy(PointType, GPU_PointType, sizeof(uchar) * rows * cols, hipMemcpyDeviceToHost);
	hipMemcpy(PointType_1D, GPU_PointType_1D, sizeof(int) * rows, hipMemcpyDeviceToHost);
	//hipMemcpy(PointType_1D, GPU_tempPointType_1D, sizeof(int) * rows, hipMemcpyDeviceToHost);
	#pragma endregion
	#pragma region Free Memory
	hipFree(GPU_DataArray);
	hipFree(GPU_PointType);
	hipFree(GPU_PointType_1D);
	hipFree(GPU_tempPointType_1D);
	#pragma endregion
	#pragma region 結束時間
	time = clock() - time;
	cout << "找最大值: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#pragma endregion
}
QImage CudaBorder::SaveDataToImage(float** DataArray)
{
	#pragma region 前置判斷
	// 要先初始化
	assert(PointType_1D != NULL && PointType != NULL  && rows != 0 && cols != 0);
	#pragma endregion
	#pragma region 開始時間
	clock_t time;
	time = clock();
	#pragma endregion
	#pragma region 透過	GPU 平行轉值
	// 原 Data Array
	float* GPU_DataArray;
	hipMalloc(&GPU_DataArray, sizeof(float) * rows * cols);
	hipMemcpy(GPU_DataArray, &DataArray[0][0], sizeof(float) * rows * cols, hipMemcpyHostToDevice);

	// Output Uint Array
	uchar *GPU_UintDataArray, *UintDataArray;
	hipMalloc(&GPU_UintDataArray, sizeof(uchar) * rows * cols);

	// 開始轉圖片
	TransforToImage << <NumBlocks, NumThreads >> > (GPU_DataArray, GPU_UintDataArray, rows, cols);
	CheckCudaError();

	// 轉成 CPU
	UintDataArray = new unsigned char[rows * cols];
	memset(UintDataArray, 0, sizeof(unsigned char) * rows * cols);
	hipMemcpy(UintDataArray, GPU_UintDataArray, sizeof(unsigned char) * rows * cols, hipMemcpyDeviceToHost);

	// 轉換到 QImage
	QImage img(UintDataArray, cols, rows, QImage::Format_Grayscale8);
	img = img.convertToFormat(QImage::Format_RGB888);
	// Debug 所有的 peak
	//for (int i = 0; i < rows * cols; i++)
	//{
	//	int rowIndex = i / cols;
	//	int colIndex = i % cols;

	//	QColor color(0, 0, 0);
	//	/*if (PointType[i] == 1)
	//		color = QColor(255, 255, 0);
	//	else */
	//	if (PointType[i] == 2)
	//	{
	//		color = QColor(255, 255, 255);
	//		img.setPixelColor(colIndex, rowIndex, color);
	//	}

	//}
	// 這邊是最後抓出來的邊界
	QColor color(255, 255, 0);
	for (int i = 0; i < rows; i++)
		if (PointType_1D[i] != -1)
			img.setPixelColor(PointType_1D[i], i, color);
	delete UintDataArray;
	hipFree(GPU_UintDataArray);

	// 判斷有無錯誤
	CheckCudaError();
	#pragma endregion
	#pragma region 結束時間
	time = clock() - time;
	cout << "轉換圖片時間: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#pragma endregion
	return img;
}

float CudaBorder::GetMaxValue(float* GPU_begin, int size)
{
	thrust::device_vector<float> d_vec(GPU_begin, GPU_begin + size);
	thrust::device_vector<float>::iterator iter = thrust::max_element(thrust::device, d_vec.begin(), d_vec.end());

	unsigned int position = iter - d_vec.begin();
	float max_val = *iter;

	CheckCudaError();

	cout << "最大值是: " << max_val << " 在位置: " << position << endl;
	return max_val;
}
void CudaBorder::NormalizeData(float *GPU_DataArray, float maxValue, int size)
{
	NormalizaDataGPU << <NumBlocks, NumThreads >> > (GPU_DataArray, maxValue, size);
	CheckCudaError();
}

//////////////////////////////////////////////////////////////////////////
// Helper Function
//////////////////////////////////////////////////////////////////////////
void CudaBorder::CheckCudaError()
{
	hipError_t GPU_Error = hipGetLastError();
	if (GPU_Error != hipSuccess)
	{
		cout << hipGetErrorString(GPU_Error) << endl;
		assert(false);
		exit(-1);
	}
}
void CudaBorder::SaveDelete(void* pointer)
{
	if (pointer != NULL)
		delete pointer;
}
