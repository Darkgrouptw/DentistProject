#include "hip/hip_runtime.h"
﻿#include "CudaBorder.cuh"

CudaBorder::CudaBorder()
{
}
CudaBorder::~CudaBorder()
{
	SaveDelete(PointType);
	SaveDelete(PointType_1D);
}

//////////////////////////////////////////////////////////////////////////
// GPU
//////////////////////////////////////////////////////////////////////////
__global__ static void NormalizaDataGPU(float* DataArray, float maxValue, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size)
		return;

	DataArray[id] /= maxValue;
}
__global__ static void findMaxAndMinPeak(float* DataArray, uchar* PointType, int size, int rows, int cols,  float MaxPeakThreshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows * cols * size)				// 超出範圍
		return;

	// width 判斷 1 ~ (width - 1)
	int colID = id % cols;
	if (1 >= colID || colID == (cols - 1))
		return;

	// 接著要去比周圍
	// 峰值判斷 (要比兩邊高，且峰值要高於某一個值，且左 或右差值，只有一端能高於這個值)
	float DiffLeft = DataArray[id] - DataArray[id - 1];
	float DiffRight = DataArray[id] - DataArray[id + 1];
	if (DiffLeft > 0 && DiffRight > 0
		&& DataArray[id] > MaxPeakThreshold)
		PointType[id] = 1;
	else if (DiffLeft < 0 && DiffRight < 0)
	//else if (DiffLeft < 0 && DiffRight < 0
	//	&& ((-DiffLeft > MinGapPeakThreshold) || (-DiffRight > MinGapPeakThreshold)))
		PointType[id] = 2;
}
__global__ static void ParseMaxMinPeak(uchar* PointType, int size, int rows, int cols, int startIndex)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)						// 超出範圍
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	// 然後這邊要去 Skip 所有的 Min
	int lastMinID = -1;
	bool FindMax = false;

	// 為了要抓出 最大(有效)的 區間
	int Useful_Start = -1;
	int Useful_End = -1;
	int Useful_PeakCount = -1, tempPeakCount = 0;

	// 刪除多餘 min Peak
	for (int i = 0; i < startIndex; i++)
	{
		int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
		PointType[currentID] = 0;
	}
	for (int i = startIndex; i < cols; i++)
	{
		int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
		if (lastMinID == -1)			// 判斷是不適剛開始 or 找到 Max
		{
			// 要先去抓出第一個 Min
			if (PointType[currentID] == 2)
				lastMinID = i;
			else if (PointType[currentID] == 1)
				PointType[currentID] = 0;				// 這邊代表沒有遇到峰值，應該是雜訊了
		}
		else
		{
			// 已經抓到 min 了之後，要去濾掉其他的 min
			if (PointType[currentID] == 1)
			{
				// 抓到 Max
				FindMax = true;
				tempPeakCount++;
			}
			else if (FindMax && PointType[currentID] == 2)
			{
				// 抓到 Max 之後，又找到一個 Min
				if (Useful_PeakCount < tempPeakCount)
				{
					Useful_PeakCount = tempPeakCount;
					Useful_Start = lastMinID;
					Useful_End = i;
				}
				FindMax = false;
				tempPeakCount = 0;
				lastMinID = -1;
			}
			else if (!FindMax && PointType[currentID] == 2)
			{
				// 沒抓到 Max 只抓到 Min
				PointType[sizeIndex * rows * cols + rowIndex * cols + lastMinID] = 0;
				lastMinID = i;
			}
		}
	}

	// 跑到最後結束，要再去判斷最後一個是否是多餘的 Min
	if (lastMinID != -1)
		PointType[sizeIndex * rows * cols + rowIndex * cols + lastMinID] = 0;
}
__global__ static void PickBestChoiceToArray(float* DataArray, uchar* PointType, int* PointType_1D, int size, int rows, int cols, int startIndex,  float Threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)							// 判斷是否超出大小
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	bool IsFindMin = false;							// 是否找到底端
	bool IsFindBorder = false;						// 是否找到邊界 (找到底端之後，要開始找邊界)
	float MinData;
	int offsetIndex = sizeIndex * size * cols + rowIndex * cols;
	for (int i = startIndex; i < cols; i++)
	{
		if (PointType[i + offsetIndex] == 2)
		{
			IsFindMin = true;
			MinData = DataArray[i + offsetIndex];
		}
		else if (IsFindMin && DataArray[i + offsetIndex] - MinData > Threshold)
		{
			IsFindBorder = true;
			PointType_1D[sizeIndex * rows + rowIndex] = i;
			break;
		}
	}

	// 接這著要判斷是否找到邊界
	// 如果沒有找到邊界，就回傳 -1
	if (!IsFindBorder)
		PointType_1D[sizeIndex * rows + rowIndex] = -1;

}
__global__ static void ConnectPointsStatus(int * PointType_1D, int* ConnectStatus, int size, int rows, int ConnectRadius)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)									// 判斷是否超出大小
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	// 代表這個點沒有有效的點
	if (PointType_1D[sizeIndex * rows + rowIndex] == -1)
		return;

	// 如果是有效的點，就繼續往下追 
	int finalPos = min(rowIndex + ConnectRadius, rows);		// 截止條件
	for (int i = rowIndex + 1; i < finalPos; i++)
	{
		if (PointType_1D[i + sizeIndex * rows] != -1)
		{
			int diffX = PointType_1D[sizeIndex * rows + rowIndex] - PointType_1D[i];
			int diffY = i - rowIndex;
			int Radius = diffX * diffX + diffY * diffY;

			// 0 沒有用到喔
			if (Radius < ConnectRadius * ConnectRadius)
			{
				int index = ConnectRadius * rowIndex + i - rowIndex + sizeIndex * rows;
				ConnectStatus[index] = Radius;
			}
		}
	}
}
__global__ static void TransforToImage(float* DataArray, uchar* OutArray, int size, int rows, int cols)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows * cols)				// 判斷是否超出大小
		return;

	float data = ((DataArray[id] / (float)3.509173f) - (float)(3.39f / 3.509173f)) * 255;
	if (data >= 255)
		OutArray[id] = 255;
	else if (data <= 0)
		OutArray[id] = 0;
	else
		OutArray[id] = (unsigned char)data;
}

//////////////////////////////////////////////////////////////////////////
// CPU
//////////////////////////////////////////////////////////////////////////
void CudaBorder::Init(int size, int rows, int cols)
{
	// 給值
	this->size = size;
	this->rows = rows;
	this->cols = cols;

	// 初始化
	SaveDelete(PointType);
	PointType = new uchar[size * rows * cols];
	memset(PointType, 0, sizeof(uchar) * size * rows * cols);
	SaveDelete(PointType_1D);
	PointType_1D = new int[size * rows];
	memset(PointType_1D, 0, sizeof(int) * size * rows);
}
void CudaBorder::GetBorderFromCuda(float* VolumeData_1D)
{
	#pragma region 前置判斷
	// 要先初始化
	assert(PointType_1D != NULL && PointType != NULL && rows != 0 && cols != 0);
	#pragma endregion
	#pragma region 開始時間
	clock_t time;
	time = clock();
	#pragma endregion
	#pragma region GPU Init
	float *GPU_VolumeData_1D;
	hipMalloc(&GPU_VolumeData_1D, sizeof(float) * size * rows * cols);
	hipMemcpy(GPU_VolumeData_1D, VolumeData_1D, sizeof(float) * size * rows * cols, hipMemcpyHostToDevice);

	// 點的型別
	uchar* GPU_PointType;
	hipMalloc(&GPU_PointType, sizeof(uchar) * size * rows * cols);
	hipMemset(GPU_PointType, 0, sizeof(uchar) * size * rows * cols);
	#pragma endregion
	#pragma region 抓取最大值 每個除以最大值
	float maxValue;
	GetMinMaxValue(VolumeData_1D, maxValue, size * rows * cols);
	NormalizaDataGPU << <NumBlocks, NumThreads >> > (GPU_VolumeData_1D, maxValue, size * rows * cols);
	CheckCudaError();

	// 找最大最小值
	findMaxAndMinPeak << < NumBlocks, NumThreads >> > (GPU_VolumeData_1D, GPU_PointType, size, rows, cols, MaxPeakThreshold);
	CheckCudaError();

	// Parse 一些連續最小值
	ParseMaxMinPeak << < NumBlocks_small, NumThreads_small >> > (GPU_PointType, size, rows, cols, StartIndex);
	CheckCudaError();

	// 抓出一維陣列
	int *GPU_PointType_1D;
	hipMalloc(&GPU_PointType_1D, sizeof(int) * size * rows);
	PickBestChoiceToArray << <NumBlocks_small, NumThreads_small >> > (GPU_VolumeData_1D, GPU_PointType, GPU_PointType_1D, size, rows, cols, StartIndex, GoThroughThreshold);
	CheckCudaError();

	// 連結點
	int *GPU_Connect_Status;
	hipMalloc(&GPU_Connect_Status, sizeof(int) * size * rows * ConnectRadius);
	hipMemset(GPU_Connect_Status, 0, sizeof(int) * size * rows * ConnectRadius);
	ConnectPointsStatus << <NumBlocks_small, NumThreads_small >> > (GPU_PointType_1D, GPU_Connect_Status, size, rows, ConnectRadius);
	CheckCudaError();

	// 把資料傳回 CPU
	int *Connect_Status = new int[size * rows * ConnectRadius];
	hipMemcpy(PointType,		GPU_PointType,		sizeof(uchar) * size * rows * cols,				hipMemcpyDeviceToHost);
	hipMemcpy(PointType_1D,	GPU_PointType_1D,	sizeof(int) * size* rows,						hipMemcpyDeviceToHost);
	hipMemcpy(Connect_Status,	GPU_Connect_Status, sizeof(int) * size * rows * ConnectRadius,		hipMemcpyDeviceToHost);

	GetLargeLine(Connect_Status);
	#pragma endregion
	#pragma region Free Memory
	hipFree(GPU_VolumeData_1D);
	hipFree(GPU_PointType);
	hipFree(GPU_PointType_1D);
	hipFree(GPU_Connect_Status);

	delete[] Connect_Status;
	#pragma endregion
	#pragma region 結束時間
	time = clock() - time;
	cout << "找邊界: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#pragma endregion
}
vector<Mat> CudaBorder::RawDataToMatArray(float* VolumeData_1D, bool SaveBorder = false)
{
	#pragma region 前置判斷
	// 要先初始化
	assert(PointType_1D != NULL && PointType != NULL && size != 0 && rows != 0 && cols != 0);
	#pragma endregion
	#pragma region 開始時間
	clock_t time;
	time = clock();
	#pragma endregion
	#pragma region 透過	GPU 平行轉值
	// 原 Data Array
	float* GPU_VolumeData_1D;
	hipMalloc(&GPU_VolumeData_1D, sizeof(float) * size * rows * cols);
	hipMemcpy(GPU_VolumeData_1D, VolumeData_1D, sizeof(float) * size * rows * cols, hipMemcpyHostToDevice);

	// Output Uint Array
	// 圖片的資料
	uchar *GPU_UintDataArray, *UintDataArray;
	hipMalloc(&GPU_UintDataArray, sizeof(uchar) * size * rows * cols);

	// 開始轉圖片
	TransforToImage << <NumBlocks, NumThreads >> > (GPU_VolumeData_1D, GPU_UintDataArray, size, rows, cols);
	CheckCudaError();

	// 轉成 CPU
	UintDataArray = new uchar[size * rows * cols];
	memset(UintDataArray, 0, sizeof(uchar) * size * rows * cols);
	hipMemcpy(UintDataArray, GPU_UintDataArray, sizeof(uchar) * size * rows * cols, hipMemcpyDeviceToHost);

	// 轉換到 Mat
	vector<Mat> ImgArray;
	for (int i = 0; i < size; i++)
	{
		// 根據 Offset 拿圖片
		Mat img(rows, cols, CV_8U, UintDataArray + i * rows * cols);
		cvtColor(img.clone(), img, CV_GRAY2BGR);

		// 丟進堆疊
		ImgArray.push_back(img);
	}

	if (SaveBorder)
	{
		// Debug 所有的 peak
		/*for (int i = 0; i < size; i++)
			for (int j = 0; j < rows * cols; j++)
			{
				int offsetIndex = i * rows * cols;
				int rowIndex = j / cols;
				int colIndex = j % cols;

				Vec3b color(0, 0, 0);
				if (PointType[offsetIndex + j] == 1)
					color = Vec3b(0, 255, 255);
				else if (PointType[offsetIndex + j] == 2)
					color = Vec3b(255, 255, 255);
				ImgArray[i].at<Vec3b>(rowIndex, colIndex) = color;
			}*/
		
		// 只抓出最後的邊界
		for (int i = 0; i < size; i++)
			for (int j = 0; j < rows; j++)
			{
				int index = i * rows + j;
				if (PointType_1D[index] != -1)
				{
					Point contourPoint(PointType_1D[index], j);
					circle(ImgArray[i], contourPoint, 2, Scalar(0, 255, 255), CV_FILLED);
				}
			}
	}
	delete[] UintDataArray;
	hipFree(GPU_UintDataArray);
	hipFree(GPU_VolumeData_1D);

	// 判斷有無錯誤
	CheckCudaError();
	#pragma endregion
	#pragma region 結束時間
	//time = clock() - time;
	//cout << "轉換圖片時間: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#pragma endregion
	return ImgArray;
}

void CudaBorder::GetMinMaxValue(float* begin, float& max, int size)
{
	clock_t time = clock();
	float* iter = max_element(begin, begin + size);
	unsigned int position = iter - begin;
	float max_val = *iter;
	//cout << "最大值是: " << max_val << " 在位置: " << position << endl;
	max = max_val;
}
void CudaBorder::GetLargeLine(int *Connect_Status)
{
	for (int i = 0; i < size; i++)
	{
		// 每個 10 段下去 Sample
		int RowGap = rows / 10;
		vector<vector<int>> StatusVector;
		for (int j = 0; j < rows; j += RowGap)
		{
			int begin = j;
			int end = j;

			if (PointType_1D[j + i * rows] == -1)
				continue;

			// 往上找 & 先加上自己
			vector<int> Connect;
			Connect.push_back(j);

			int FindIndex = j;
			bool IsFind = true;
			while (IsFind && FindIndex > 0)
			{
				int minIndex = -1;
				int tempValue = ConnectRadius * ConnectRadius;
				for (int k = 1; k < ConnectRadius; k++)
				{
					int index = ConnectRadius * (FindIndex - k) + k + i * rows * ConnectRadius;
					if (FindIndex - k >= 0 && Connect_Status[index] != 0 && tempValue > Connect_Status[index])
					{
						tempValue = Connect_Status[index];
						minIndex = k;
					}
				}

				if (minIndex != -1)
				{
					FindIndex = FindIndex - minIndex;
					Connect.push_back(FindIndex);
					IsFind = true;
				}
				else
					IsFind = false;
			}

			// 往下找
			FindIndex = j;
			while (IsFind && FindIndex < rows - 1)
			{
				int minIndex = -1;
				int tempValue = ConnectRadius * ConnectRadius;
				for (int k = 1; k < ConnectRadius; k++)
				{
					int index = ConnectRadius * FindIndex + k + i * rows * ConnectRadius;
					if (FindIndex + k < rows && Connect_Status[index] != 0 && tempValue > Connect_Status[index])
					{
						tempValue = Connect_Status[index];
						minIndex = k;
					}
				}

				if (minIndex != -1)
				{
					FindIndex = FindIndex + minIndex;
					Connect.push_back(FindIndex);
					IsFind = true;
				}
				else
					IsFind = false;
			}

			if (Connect.size() > 1)
			{
				// 由小排到大
				sort(Connect.begin(), Connect.end());
				StatusVector.push_back(Connect);
			}
		}

		// 前面的幾個張數，可能會找不到點，所以跳過處理
		if (StatusVector.size() == 0)
			continue;

		// 排序之後取最大
		sort(StatusVector.begin(), StatusVector.end(), SortByVectorSize);

		// 把其他雜點刪掉
		vector<int> LineVector = StatusVector[0];
		int index = 0;
		for (int j = 0; j < rows; j++)
		{
			if (LineVector[index] != i)
				PointType_1D[j + i * rows] = -1;
			else if (LineVector[index] == i)
			{
				index++;
				if (index >= LineVector.size())
				{
					for (int k = j + 1; k < rows; k++)
						PointType_1D[k + i * rows] = -1;
					break;
				}
			}
		}
	}
}
bool CudaBorder::SortByVectorSize(vector<int> left, vector<int> right)
{
	return right.size() < left.size();
}

//////////////////////////////////////////////////////////////////////////
// Helper Function
//////////////////////////////////////////////////////////////////////////
void CudaBorder::CheckCudaError()
{
	hipError_t GPU_Error = hipGetLastError();
	if (GPU_Error != hipSuccess)
	{
		cout << hipGetErrorString(GPU_Error) << endl;
		assert(false);
		exit(-1);
	}
}
void CudaBorder::SaveDelete(void* pointer)
{
	if (pointer != NULL)
		delete[] pointer;
}
