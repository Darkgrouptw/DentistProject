#include "hip/hip_runtime.h"
﻿#include "TRCudaV2.cuh"
#include "EigenUtility.h"

TRCudaV2::TRCudaV2()
{
}
TRCudaV2::~TRCudaV2()
{
	// 這邊要刪除其他創出來的變數
	SaveDelete(VolumeData);
	SaveDelete(PointType);
	SaveDelete(PointType_1D);
}

//////////////////////////////////////////////////////////////////////////
// GPU
//////////////////////////////////////////////////////////////////////////
__device__ static float Z1Function(float x1)
{
	// 這個 Function 不確定在幹嘛XD
	// https://i.imgur.com/QS3bczf.png
	return -126.4517 + 
		0.4005123 *				x1 -
		0.000011981 *			pow(x1 - 2122.41, 2) -
		0.000000011664 *		pow(x1 - 2122.41, 3) +
		0.000000000001432 *		pow(x1 - 2122.41, 4) -
		0.0000000000000008164 * pow(x1 - 2122.41, 5) +
		5.939E-20 *				pow(x1 - 2122.41, 6);
}
__global__ static void RawDataToOriginalData(char* FileRawData, int* OCTRawData, int OCTDataSize)
{
	// 這邊是原本讀取是 1個 Byte 要轉乘 2個 Bytes 為一筆資料
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 這邊應該是不會發生，就當作例外判斷
	if (id >= OCTDataSize)
	{
		printf("轉 Raw Data 有 Error!\n");
		return;
	}

	OCTRawData[id] = (int)((uchar)FileRawData[id * 2]) + (int)((uchar)FileRawData[id * 2 + 1]) * 256;
}
__global__ static void CombineTwoChannels_Single(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
	// 這邊是 Denoise，把兩個 Channel 的資料相加
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 這邊應該是不會發生，就當作例外判斷
	if (id >= SizeX * SizeY * SizeZ)
	{
		printf("Combine Two Channel 有 Error!\n");
		return;
	}

	int BoxSize = SizeX * SizeZ;										// 這邊沒有反掃，所以直接接上大小
	int BoxIndex = id / BoxSize;
	int BoxLeft = id % BoxSize;

	OCTData[id] = (OCTData_2Channls[BoxIndex * 2 * BoxSize + BoxLeft] +
		OCTData_2Channls[(BoxIndex * 2 + 1) * BoxSize + BoxLeft]) / 2;
}
__global__ static void CombineTwoChannels_Multi(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
	// 這邊是 Denoise，把兩個 Channel 的資料相加
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 這邊應該是不會發生，就當作例外判斷
	if (id >= SizeX * SizeY * SizeZ)
	{
		printf("Combine Two Channel 有 Error!\n");
		return;
	}

	int BoxSize = SizeX * SizeZ * 2;									// 一個 Channel 的資料是 正掃 + 反掃
	int BoxIndex = id / BoxSize;
	int BoxLeft = id % BoxSize;

	OCTData[id] = (OCTData_2Channls[BoxIndex * 2 * BoxSize + BoxLeft] +
				OCTData_2Channls[(BoxIndex * 2 + 1) * BoxSize + BoxLeft]) / 2;
}
__global__ static void ReverseBackScanData(int* OCTData, int SizeX, int SizeY, int SizeZ)
{
	// 這邊是要反轉 反掃的資料
	int id = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +			// Y	=> (Y * 2 + 1) * (2 * 1024)						=> 1, 3, 5, 7, 9
		blockIdx.x * gridDim.z * blockDim.x +											// X	=> X * (125 * 2) * (2 * 1024)
		blockIdx.z * blockDim.x +														// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	int changeID = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +		// Y	=> (Y * 2 + 1) * (2 * 1024)						=> 1, 3, 5, 7, 9
		(gridDim.y * 2 - blockIdx.x - 1) * gridDim.z * blockDim.x +						// X	=> (250 - X - 1) * (125 * 2) * (2 * 1024)
		blockIdx.z * blockDim.x +														// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	int value = OCTData[id];
	OCTData[id] = OCTData[changeID];
	OCTData[changeID] = value;
}
__global__ static void GetMatrixA(int* OCTData, float* MatrixA, int NumPolynomial, int OneDataSize)
{
	// 這個 Function 是去取得 MatrixA 的值
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// 例外判斷 (理論上應該也是不會超過)
	if (id >= (NumPolynomial + 1) * (NumPolynomial + 1))
	{
		printf("多項式 Fitting 有問題!\n");
		return;
	}

	// 算 Index
	int rowIndex = id % (NumPolynomial + 1);
	int colsIndex = id / (NumPolynomial + 1);

	// 做相加
	float value = 0;
	for (int i = 0; i < OneDataSize; i++)
	{
		// 抓出兩項的值
		float FirstValue = (float)i / OneDataSize;
		float SecondValue = (float)i / OneDataSize;
		value += pow(FirstValue, NumPolynomial - rowIndex) * pow(SecondValue, NumPolynomial - colsIndex);
	}
	MatrixA[id] = value;
}
__global__ static void GetMatrixB(int* OCTData, float* MatrixB, float YAverage, int NumPolynomial, int OneDataSize)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// 算 Index
	int rowIndex = id % (NumPolynomial + 1);
	int colsIndex = id / (NumPolynomial + 1);

	// 做相加
	float value = 0;
	for (int i = 0; i < OneDataSize; i++)
	{
		// 抓出兩項的值
		float FirstValue = (float)i / OneDataSize;
		float SecondValue = OCTData[i] - YAverage;
		value += pow(FirstValue, NumPolynomial - rowIndex) * SecondValue;
	}
	MatrixB[id] = value;
}
__global__ static void MinusByFittingFunction(int* OCTData, float* PolyValue, int SizeZ)
{	
	// 這邊要減掉 Fitting Data
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 先拿出他是第幾個 Z
	int idZ = id % SizeZ;

	// 減掉預測的值
	OCTData[id] -= PolyValue[idZ];
}
__global__ static void ComputePXScale(float* PXScale, int OffsetBegin, int ShiftValue, int Steps, int Size)
{
	// 這邊是算出 PXScale Array(詳細在幹嘛我不是很懂@@)
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= Size)
	{
		printf("ComputePXScale 有問題!\n");
		return;
	}

	// 聽說是去直流
	int idOffset = OffsetBegin + ShiftValue;
	PXScale[id] = (Z1Function(idOffset + id) - Z1Function(idOffset)) * Steps;
}
__global__ static void FrequencyAdjust(int* OCTData, float* KSpaceData, float* PXScale, int* IndexArray, int CutIndex, int SizeX, int SizeY, int SizeZ)
{
	// 這邊是 Denoise，把兩個 Channel 的資料相加
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	if (id >= SizeX * SizeY * SizeZ)
	{
		printf("Frequency 轉換的地方有問題");
		return;
	}

	// 算回原本的 Index
	int idZ = id % SizeZ;
	if (IndexArray[idZ] == -1 || idZ >= CutIndex || idZ == 0)
	{
		KSpaceData[id] = 0;
		return;
	}

	// 要算斜率前，先拿出上一筆資料
	int LastPXScaleIndex = (IndexArray[idZ] - 1 <= 0 ? 0 : IndexArray[idZ] - 1);

	double m = (double)(OCTData[id] - OCTData[id - 1]) / (PXScale[IndexArray[idZ]] - PXScale[LastPXScaleIndex]);
	double c = OCTData[id] - m * PXScale[IndexArray[idZ]];
	KSpaceData[id] = m * idZ + c;
}
__global__ static void DataToComplexData(float* KSpaceData, hipfftComplex* FFTData, int OCTDataSize)
{	
	// 把 KSpace 的 Data 塞進 FFT
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	if (id >= OCTDataSize)
	{
		printf("放進 Complex Data 有錯誤!!\n");
		return;
	}

	// 放進 Complex Data 裡
	FFTData[id].x = KSpaceData[id];
	FFTData[id].y = 0;
}
__global__ static void ComplexDataToData(hipfftComplex* FFTData, float* OCTFloatData, int SizeX, int SizeY, int SizeZ, int OCTDataSize)
{
	// FFT 資料塞回原本的資料集
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (1 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (0 * 1024 + Z2)
		threadIdx.x;

	if (id >= OCTDataSize / 2)
	{
		printf("Complex To Data 有錯誤!!\n");
		return;
	}

	// 這邊要避免 0 頻率與 最大頻率(由於只取一半的右邊，所以只拿 1024)，詳情請看 Youtube 連結 (你看學長有多好，都找連結給你了，還不看!!)
	// 這邊要除以 2 是因為它會對稱
	// 然後拿的順序要反過來 (由於東元那邊的程式是這樣)
	// 如果是最大頻率 (也就是 Size / 2 - 1 => 1023)，那就要去下一個 也就是 1022 
	/*int idZ = id % (SizeZ / 2);
	idZ = SizeZ / 2 - idZ - 1;
	if (idZ == SizeZ / 2 - 1)
		idZ--;*/
	int idZ = id % (SizeZ / 2);
	if (idZ == 0)
		idZ++;
	

	// 這邊的算法要對應回去原本的資料
	int tempIndex = id / (SizeZ / 2);
	int idX = tempIndex % SizeX;
	int idY = tempIndex / SizeX;
	int NewIndex = idY * SizeX * SizeZ + idX * SizeZ + idZ;
	float temp = sqrt(FFTData[NewIndex].x * FFTData[NewIndex].x + FFTData[NewIndex].y * FFTData[NewIndex].y);

	// 做一下例外判斷
	if (temp == 0)
		OCTFloatData[id] = 0;
	else
		OCTFloatData[id] = log10f(temp) * 10;
}
__global__ static void ShiftFinalData(float* AfterFFTData, float* ShiftData, int SizeX, int SizeY, int FinalSizeZ, int FinalDataSize)
{
	// 這邊要做位移
	// 由於硬體是這樣子 ↓
	// => | ->
	// ("->" 是指第一段，"=>" 是指第二段)
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;
	
	if (id >= FinalDataSize)
	{
		printf("Shift Data 有錯誤!!\n");
		return;
	}
	
	// 這邊的算法要對應回去原本的資料
	int idZ = id % FinalSizeZ;
	int tempIndex = id / FinalSizeZ;
	int idX = tempIndex % SizeX;
	int idY = tempIndex / SizeX;

	// SizeY 折回來
	// (0 ~ 124 125 ~ 249)
	//		↓
	// (125 ~ 249 0 ~ 124)
	idY = (idY + SizeY / 2) % SizeY;

	int NewIndex = idY * SizeX * FinalSizeZ + idX * FinalSizeZ + idZ;
	ShiftData[id] = AfterFFTData[NewIndex];
	//ShiftData[id] = AfterFFTData[id];
}
__global__ static void GpuDataToCpuData(float* GpuData, float* CpuData, int FinalDataSize) {
	// 這邊是根據資料的最大最小值，去做 Normalize 資料
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 例外判斷
	if (id >= FinalDataSize)
	{
		printf("Move Data 超出範圍\n");
		return;
	}

	CpuData[id] = GpuData[id];
}
__global__ static void NormalizeData(float* ShiftData, float MaxValue, float MinValue, int FinalDataSize)
{
	// 這邊是根據資料的最大最小值，去做 Normalize 資料
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * (2 * 1024)
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * (2 * 1024)
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	// 例外判斷
	if (id >= FinalDataSize)
	{
		printf("Normaliza Data 超出範圍\n");
		return;
	}

	if (ShiftData[id] < MinValue)
		ShiftData[id] = 0;
	else if (ShiftData[id] > MaxValue)
		ShiftData[id] = 1;
	else
		ShiftData[id] = (ShiftData[id] - MinValue) / (MaxValue - MinValue);

}

// 轉成圖片 & 產生邊界判斷 (Smooth 後的 Data)的資料
__device__ static float SmoothDataByIndex(float* VolumeData, int id, int FinalSizeZ, int SmoothSizeRange)
{
	int idZ = id % FinalSizeZ;
	int SmoothRadius = (SmoothSizeRange - 1) / 2;

	// Smooth 這個區段的資料
	int MinValue = min(SmoothRadius, idZ - 0);
	int MaxValue = min(SmoothRadius, FinalSizeZ - idZ - 1);
	float TempTotal = 0;

	// 把範圍內的部分相加
	for (int i = -MinValue; i <= MaxValue; i++)
		TempTotal += VolumeData[id + i];
	TempTotal /= (MaxValue + MinValue + 1);
	return TempTotal;
}
__global__ static void TransformToImageAndBorderData(float* VolumeData_Normalized, float* SmoothData, uchar* ImageArray, int SizeX, int SizeY, int FinalSizeZ, int SmoothSizeRange)
{
	// 這邊是將原本的資料，轉換完圖片
	int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +			// Y	=> Y * 250 * 1 * 1024
		blockIdx.x * gridDim.z * blockDim.x +							// X	=> X * 1 * 1024
		blockIdx.z * blockDim.x +										// Z	=> (Z1 * 1024 + Z2)
		threadIdx.x;

	if (id >= SizeX * SizeY * FinalSizeZ)								// 判斷是否超出大小
		return;

	// 產生 Border Detect 的資料
	SmoothData[id] = SmoothDataByIndex(VolumeData_Normalized, id, FinalSizeZ, SmoothSizeRange);

	// 這個 1.3 倍，是東元測出來的
	float data = VolumeData_Normalized[id] * 255 * 1.3f;
	if (data >= 255)
		ImageArray[id] = 255;
	else if (data <= 0)
		ImageArray[id] = 0;
	else
		ImageArray[id] = (uchar)data;
}

// 邊界部分
__global__ static void ZCalcBrightness(float* DataArray, float* BrightArray, int size, int rows, int cols, int startIndex)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)						// 超出範圍
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	BrightArray[id] = 0;
	for (int i = startIndex; i < cols; i++)
	{
		int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
		BrightArray[id] += DataArray[currentID];
	}
}
__global__ static void findMaxAndMinPeak(float* DataArray, float* BrightnessArray, uchar* PointType, int size, int rows, int cols, float MaxPeakThreshold, float SatPeakThreshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= rows * cols * size)				// 超出範圍
		return;

	// width 判斷 1 ~ (width - 1)
	int colID = id % cols;
	if (1 >= colID || colID == (cols - 1))
		return;

	// 是否飽和
	int tempIndex = id / cols;
	if (BrightnessArray[tempIndex] > SatPeakThreshold)
		return;

	// 接著要去比周圍
	// 峰值判斷 (要比兩邊高，且峰值要高於某一個值，且左 或右差值，只有一端能高於這個值)
	float DiffLeft = DataArray[id] - DataArray[id - 1];
	float DiffRight = DataArray[id] - DataArray[id + 1];
	if (DiffLeft > 0 && DiffRight > 0
		&& DataArray[id] > MaxPeakThreshold)
		PointType[id] = 1;
	else if (DiffLeft < 0 && DiffRight < 0)
		PointType[id] = 2;
}
__global__ static void ParseMaxMinPeak(uchar* PointType, int size, int rows, int cols, int startIndex)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)						// 超出範圍
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	// 然後這邊要去 Skip 所有的 Min
	int lastMinID = -1;
	bool FindMax = false;

	// 為了要抓出 最大(有效)的 區間
	int Useful_Start = -1;
	int Useful_End = -1;
	int Useful_PeakCount = -1, tempPeakCount = 0;

	// 刪除多餘 min Peak
	for (int i = 0; i < startIndex; i++)
	{
		int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
		PointType[currentID] = 0;
	}
	for (int i = startIndex; i < cols; i++)
	{
		int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
		if (lastMinID == -1)			// 判斷是不適剛開始 or 找到 Max
		{
			// 要先去抓出第一個 Min
			if (PointType[currentID] == 2)
				lastMinID = i;
			else if (PointType[currentID] == 1)
				PointType[currentID] = 0;				// 這邊代表沒有遇到峰值，應該是雜訊了
		}
		else
		{
			// 已經抓到 min 了之後，要去濾掉其他的 min
			if (PointType[currentID] == 1)
			{
				// 抓到 Max
				FindMax = true;
				tempPeakCount++;
			}
			else if (FindMax && PointType[currentID] == 2)
			{
				// 抓到 Max 之後，又找到一個 Min
				if (Useful_PeakCount < tempPeakCount)
				{
					Useful_PeakCount = tempPeakCount;
					Useful_Start = lastMinID;
					Useful_End = i;
				}
				FindMax = false;
				tempPeakCount = 0;
				lastMinID = -1;
			}
			else if (!FindMax && PointType[currentID] == 2)
			{
				// 沒抓到 Max 只抓到 Min
				PointType[sizeIndex * rows * cols + rowIndex * cols + lastMinID] = 0;
				lastMinID = i;
			}
		}
	}

	// 跑到最後結束，要再去判斷最後一個是否是多餘的 Min
	if (lastMinID != -1)
		PointType[sizeIndex * rows * cols + rowIndex * cols + lastMinID] = 0;
}
__device__ static void InsertBestNChoice(float* CandidateGap, int* PointType_BestN, int offsetIndex, int bestNoffsetIndex, int CurrentIndex, int ChooseBestN)
{
	bool IsInsert = false;
	for (int i = 0; i < ChooseBestN && !IsInsert; i++)
	{
		// 大於 0 代表已經有值了
		if (PointType_BestN[bestNoffsetIndex + i] > 0)
		{
			// 比較
			int preIndex = PointType_BestN[bestNoffsetIndex + i];
			if (CandidateGap[offsetIndex + preIndex] >= CandidateGap[offsetIndex + CurrentIndex])		// 原先的比他大，代表不加入，找下一個
				continue;
			else if (CandidateGap[offsetIndex + preIndex] < CandidateGap[offsetIndex + CurrentIndex])	// 把剩下來的往後推，並加入此答案
			{
				for (int j = ChooseBestN - 1; j > i; j--)
					PointType_BestN[bestNoffsetIndex + j] = PointType_BestN[bestNoffsetIndex + j - 1];
				PointType_BestN[bestNoffsetIndex + i] = CurrentIndex;
				IsInsert = true;
			}
		}
		else
		{
			PointType_BestN[bestNoffsetIndex + i] = CurrentIndex;
			break;
		}
	}
}
__global__ static void PickBestChoiceToArray(float* DataArray, uchar* PointType, float* CandidateGap, int* PointType_BestN, int size, int rows, int cols, int ChooseBestN, int startIndex, float Threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)							// 判斷是否超出大小
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	bool IsFindMin = false;											// 是否找到底端
	float MinData;
	int offsetIndex = sizeIndex * rows * cols + rowIndex * cols;
	int bestNoffsetIndex = sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN;
	float lastData = -1;
	for (int i = startIndex; i < cols; i++)
	{
		// 先找最小的
		if (PointType[i + offsetIndex] == 2)
		{
			// 如果前面已經有找到其他點的話
			if (IsFindMin)
				lastData = -1;

			IsFindMin = true;
			MinData = DataArray[i + offsetIndex];
		}
		else if (
			IsFindMin &&											// 要先找到最低點
			PointType[i + offsetIndex] == 1 &&
			DataArray[i + offsetIndex] - MinData > Threshold		// 接著找大於這個 Threshold
			)
		{
			lastData = DataArray[i + offsetIndex] - MinData;

			// 把差距加進去，跟前面的比較，找出最好的加入 PointType_BestN
			CandidateGap[offsetIndex + i] = lastData;
			InsertBestNChoice(CandidateGap, PointType_BestN, offsetIndex, bestNoffsetIndex, i, ChooseBestN);
		}
	}

	// 把其他的設定為 0
	for (int i = 0; i < ChooseBestN; i++)
		if (PointType_BestN[bestNoffsetIndex + i] == 0)
			PointType_BestN[bestNoffsetIndex + i] = -1;
}
__global__ static void CalcNeighbor(int* PointType_BestN, float* NeighborCountArray, int size, int rows, int cols, int ChooseBestN, int Radius)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows)							// 判斷是否超出大小
		return;

	// 算 Index
	int sizeIndex = id / rows;
	int rowIndex = id % rows;

	// 先塞 index
	int chooseIndex = sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN;
	for (int i = 0; i < ChooseBestN; i++)
	{
		// 清空陣列
		int totalPixelCount = 0;
		float avgPixel = 0;

		int BestN = PointType_BestN[chooseIndex + i];
		if (BestN == -1)
		{
			NeighborCountArray[chooseIndex + i] == 0;
			continue;
		}

		// 算有幾個在鄰居
		for (int y = -Radius; y <= Radius; y++)
			for (int x = -Radius; x <= Radius; x++)
				for (int n = 0; n < ChooseBestN; n++)
				{
					int currentSizeIndex = sizeIndex + y;
					int currentRowIndex = rowIndex + x;
					if (0 <= currentSizeIndex && currentSizeIndex < size &&
						0 <= currentRowIndex && currentRowIndex < rows)
					{
						totalPixelCount++;
						int CurrentBestNIndex = currentSizeIndex * rows * ChooseBestN + currentRowIndex * ChooseBestN + n;
						int CurrentBestN = PointType_BestN[CurrentBestNIndex];
						
						// 如果沒有東西就 Return
						if (CurrentBestN == -1)
							continue;

						if (abs(CurrentBestN - BestN) <= Radius)
							avgPixel++;
					}
				}

		// 算完之後，先塞到裡面
		NeighborCountArray[chooseIndex + i] = avgPixel / totalPixelCount;
	}

	// 只保留最大的
	int maxIndex = (thrust::max_element(thrust::device, NeighborCountArray + chooseIndex, NeighborCountArray + chooseIndex + ChooseBestN) - (NeighborCountArray + chooseIndex));
	PointType_BestN[chooseIndex] = PointType_BestN[chooseIndex + maxIndex];
	for (int i = 1; i < ChooseBestN; i++)
		PointType_BestN[i] = -1;
}
__global__ static void ConnectPointsStatus(int* PointType_BestN, int* ConnectStatus, int size, int rows, int ChooseBestN, int ConnectRadius)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= size * rows * ChooseBestN)						// 判斷是否超出大小
		return;

	// 算 Index
	int sizeIndex = id / (rows * ChooseBestN);
	int tempID = id % (rows * ChooseBestN);
	int rowIndex = tempID / ChooseBestN;
	int chooseIndex = tempID % ChooseBestN;

	// 代表這個點沒有有效的點
	if (PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] == -1)
		return;

	// 如果是有效的點，就繼續往下追 
	int finalPos = min(rowIndex + ConnectRadius, rows);		// 截止條件
	for (int i = rowIndex + 1; i < finalPos; i++)
	{
		for (int j = 0; j < ChooseBestN; j++)
		{
			// 下一個點的位置 (第 i 個 row 的點)
			// 然後的第 1 個點
			if (PointType_BestN[sizeIndex * rows * ChooseBestN + i * ChooseBestN + j] != -1)
			{
				// 前面項為現在這個點
				// 後面項為往下的點
				int diffX = PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] -
					PointType_BestN[sizeIndex * rows * ChooseBestN + i * ChooseBestN + j];
				int diffY = i - rowIndex;
				int Radius = diffX * diffX + diffY * diffY;

				// 0 沒有用到喔
				if (Radius < ConnectRadius * ConnectRadius)
				{
					// 張數的位移 + Row 的位移 + 現在在 Top N 的點 + 半徑的位移 + 往下 Top N 的結果
					int index = sizeIndex * rows * ChooseBestN * ConnectRadius * ChooseBestN +			// 張數
						rowIndex * ChooseBestN * ConnectRadius * ChooseBestN +					// Row
						chooseIndex * ConnectRadius * ChooseBestN +								// 現在在 Top N 
						(i - rowIndex) * ChooseBestN +											// 半徑
						j;
					ConnectStatus[index] = Radius;
				}
			}
		}
	}
}

// 這邊是例外，只有 Multi 才有TopView 
__global__ static void GetOtherSideView(float* Data, float* OtherSideData, int SizeX, int SizeY, int FinalSizeZ)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= SizeX * SizeY)
	{
		printf("範圍有錯!!\n");
		return;
	}

	// id 換算
	int idX = id / SizeY;
	int idY = id % SizeY;
	int DataOffsetIndex = idX * SizeY * FinalSizeZ + idY * FinalSizeZ;

	// 總和一個 SizeZ
	float totalZ = 0;
	for (int i = 0; i < FinalSizeZ; i++)
		totalZ += Data[DataOffsetIndex + i];


	// 這邊的單位要調整一下
	// rows => 是張樹 (SizeY)
	// cols => 是 SizeX
	int offsetIndex = idY * SizeX + idX;
	OtherSideData[offsetIndex] = totalZ;
}
__global__ static void TransformOtherSideDataToImage(float* OtherSideData, uchar* UintOtherSideData, float Mean, float FixMean, int SizeX, int SizeY)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= SizeX * SizeY)								// 判斷是否超出大小
		return;

	// 位移到設定的 Mean 直間
	float ScaleFactor = FixMean / Mean / 255;
	float data = OtherSideData[id] * 255 * ScaleFactor;
	if (data >= 255)
		UintOtherSideData[id] = 255;
	else if (data <= 0)
		UintOtherSideData[id] = 0;
	else
		UintOtherSideData[id] = (uchar)data;
}

//////////////////////////////////////////////////////////////////////////
// CPU
//////////////////////////////////////////////////////////////////////////
// 轉換 Function
void TRCudaV2::SingleRawDataToPointCloud(char* FileRawData, int DataSize, int SizeX, int SizeZ, long ShiftValue, double K_Step, int CutValue)
{
	// 算時間
	#ifdef SHOW_TRCUDAV2_TOTAL_TIME
	totalTime = clock();
	#endif

	//////////////////////////////////////////////////////////////////////////
	// 步驟說明
	// 1. 上傳 GPU Data
	// 2. 一開始要把資料讀進來 (由於原本的資料都是 2個 Bytes 為一組，但 QT 目前是先用 GPU 轉換到 2個 Bytes)，和
	//    由於資料有 兩個 Channels，要相加除以2，可以去除雜訊 (由於原本的能量強度資料是使用三角波，所以會有去跟回兩個資料，就是把這兩筆資料相加除以 2)
	// 3. 用 5 次項去 Fit 一條曲線
	// 4. λ Space 轉成 K Space
	// 5. cuFFT
	// (這個部分不用位移)
	// 7. 根據最大最小值來 Normalize 資料
	// 8. 轉成圖
	// 9. 邊界判斷
	// 10. 抓下 GPU Data
	//
	// 細節說明：
	// 1. 轉換 Function => X 快軸、Y 慢軸
	// 2. ShiftValue	=> TRIGGER DELAY位移(換FIBER，電線校正回來用的)
	// 3. K_Step		=> 深度(14.多mm對應 2.5的k step；可以考慮之後用2)(k step越大，z軸越深，但資料精細度越差；1~2.5)
	// 4. CutValue		=> OCT每個z軸，前面數據減去多少。原因是開頭的laser弱，干涉訊號不明顯，拿掉的資料會比較美。 (東元那邊的變數是 cuteValue XD)
	// 5. 這邊如果是 2 Channel 的話，大小為 2048	x	250		x	2				x 2			x 2               
	//										(深度)	x	(快軸)	x	(慢軸(反掃))	x Channel	x 2個 Byte 為一組
	//////////////////////////////////////////////////////////////////////////
	#pragma region 1. 上傳 GPU Data
	// 初始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	clock_t time = clock();
	#endif

	// GPU Data
	char* GPU_FileRawData;			// => 從檔案讀進來的 Raw Data
	int *GPU_OCTRawData_2Channel;	// => 這個是 OCT 掃完全部的 Raw Data (2Channels，如果"只用到一個" Channel 那就不會用到這個陣列)
	int *GPU_OCTRawData;			// => 這個是實際 Denoise 的 Data (也就是 CH1 + CH2 的資料) (如果"只有一個" Channel，就只會用到這個陣列)
	float *GPU_OCTFloatData;		// => 這個會用在兩個地方，一個是 K Space 的資料，一個是 FFT 後的資料

	// 注意!! 因為只拿一組，不需要 兩個慢軸的資訊 (也就是反掃的資訊)，所以除以 2
	DataSize /= 2;

	// 是否是 2 Channels
	bool UseTwoChannels = (DataSize / SizeX / SizeZ == 4);		// 2 Byte & 2 Channles

	// 原始資料
	hipMalloc(&GPU_FileRawData, sizeof(char) * DataSize);

	// 這邊要分兩個 Copy (略過反掃資料)
	hipMemcpy(GPU_FileRawData,					FileRawData,			sizeof(char) * DataSize / 2, hipMemcpyHostToDevice);
	hipMemcpy(GPU_FileRawData + DataSize / 2,	FileRawData + DataSize,	sizeof(char) * DataSize / 2, hipMemcpyHostToDevice);
	CheckCudaError();

	// 判對是否使用 2 Chanels
	int OCTDataSize = SizeX * SizeZ;
	if (UseTwoChannels)
		hipMalloc(&GPU_OCTRawData_2Channel, sizeof(int) * OCTDataSize * 2);
	hipMalloc(&GPU_OCTRawData, sizeof(int) * OCTDataSize);
	hipMalloc(&GPU_OCTFloatData, sizeof(float) * OCTDataSize);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "1. 上傳至 GPU: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion 
	#pragma region 2. 讀檔轉換
	//////////////////////////////////////////////////////////////////////////
	// 這邊的資料格式是這樣
	// ↗↘↗↘ 是一組 (↗代表掃描 0 ~ 250的一次資料)
	// 其中一個 ↗↘ 是一個三角波的資料
	// 但因為有兩個 channel 所以一組資料是 ↗↘↗↘
	//////////////////////////////////////////////////////////////////////////
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 解出 2 Byte 的資料
	if (UseTwoChannels)
	{
		RawDataToOriginalData << < dim3(SizeX, 1, SizeZ / NumThreads * 2), NumThreads >> > (GPU_FileRawData, GPU_OCTRawData_2Channel, DataSize / 2);
		CheckCudaError();

		// 兩個 Channel 作 Denoise
		CombineTwoChannels_Single << < dim3(SizeX, 1, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData_2Channel, GPU_OCTRawData, SizeX, 1, SizeZ);

		// 刪除
		hipFree(GPU_OCTRawData_2Channel);
	}
	else
		RawDataToOriginalData << < dim3(SizeX, 1, SizeZ / NumThreads), NumThreads >> > (GPU_FileRawData, GPU_OCTRawData, DataSize / 2);
	CheckCudaError();

	// 刪除 FileRaw Data
	hipFree(GPU_FileRawData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "2. 讀檔轉換: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 3. 用五次項去 Fitting
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 初始化 Matrix
	float* GPU_MatrixA;
	float* GPU_MatrixB;
	hipMalloc(&GPU_MatrixA, sizeof(float) * (NumPolynomial + 1) *(NumPolynomial + 1));
	hipMalloc(&GPU_MatrixB, sizeof(float) * (NumPolynomial + 1));

	// 先算平均
	int* FirstSizeZData = new int[SizeZ];
	memset(FirstSizeZData, 0, sizeof(int) * SizeZ);
	hipMemcpy(FirstSizeZData, GPU_OCTRawData, sizeof(int) * SizeZ, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	float average = accumulate(FirstSizeZData, FirstSizeZData + SizeZ, 0.0f) / SizeZ;
	delete[] FirstSizeZData;

	// 取得 Matrix
	GetMatrixA << <1, (NumPolynomial + 1) * (NumPolynomial + 1) >> > (GPU_OCTRawData, GPU_MatrixA, NumPolynomial, SizeZ);
	GetMatrixB << <1, NumPolynomial + 1 >> > (GPU_OCTRawData, GPU_MatrixB, average, NumPolynomial, SizeZ);
	CheckCudaError();

	float* MatrixA = new float[(NumPolynomial + 1) *(NumPolynomial + 1)];
	float* MatrixB = new float[(NumPolynomial + 1)];
	hipMemcpy(MatrixA, GPU_MatrixA, sizeof(float) * (NumPolynomial + 1) *(NumPolynomial + 1), hipMemcpyDeviceToHost);
	hipMemcpy(MatrixB, GPU_MatrixB, sizeof(float) * (NumPolynomial + 1), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// 解 Eigen 找 Fitting Function
	EigenUtility eigen;
	eigen.SetAverageValue(average);
	eigen.SolveByEigen(MatrixA, MatrixB, NumPolynomial);
	
	// 扣除那個 Function
	float* GPU_PolyValue;
	float* PolyValue = eigen.GetFunctionArray(SizeZ, average);
	hipMalloc(&GPU_PolyValue, sizeof(float) * SizeZ);
	hipMemcpy(GPU_PolyValue, PolyValue, sizeof(float) * SizeZ, hipMemcpyHostToDevice);
	MinusByFittingFunction << < dim3(SizeX, 1, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData, GPU_PolyValue, SizeZ);
	CheckCudaError();

	// 刪除多出來的
	hipFree(GPU_MatrixA);
	hipFree(GPU_MatrixB);
	hipFree(GPU_PolyValue);
	delete[] MatrixA;
	delete[] MatrixB;
	delete[] PolyValue;

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "3. 多項式去 Fitting : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 4. λ Space 轉成 K Space
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 初始化
	float* PX_Scale = new float[SizeZ];
	int* KSpaceIndexArray = new int[SizeZ];
	float* GPU_PXScale;
	int* GPU_KSpaceIndexArray;
	hipMalloc(&GPU_PXScale,			sizeof(float) * SizeZ);
	hipMalloc(&GPU_KSpaceIndexArray,	sizeof(int) * SizeZ);

	// 設定一些系數
	int OffsetBegin = 800;

	// 算出 PXScale 的 Array
	ComputePXScale << <SizeZ / NumThreads, NumThreads >> > (GPU_PXScale, OffsetBegin, ShiftValue, K_Step, SizeZ);
	CheckCudaError();

	// 抓下來準備算 K Space Index (由於這邊如果使用 GPU 去做，會導致大部分的 thread 在等最大工作量的 thread，所以這裡 CPU 做會比較快)
	hipMemcpy(PX_Scale, GPU_PXScale, sizeof(float) * SizeZ, hipMemcpyDeviceToHost);

	// 算 K Space 的對應 Array
	int index = 1;
	int KSpaceOffset = PX_Scale[SizeZ - 1];
	for (int i = 0; i <= KSpaceOffset; i++)
	{
		while (i >= PX_Scale[index])
		{
			index++;
		}
		KSpaceIndexArray[i] = index;
	}
	for (int i = KSpaceOffset + 1; i < SizeZ; i++)
		KSpaceIndexArray[i] = -1;

	// 由於 K Space 不是線性關係，所以要從 KSpaceIndexArray，找 Index，再從左右兩個點中，內插出實際在這個 Index 的值
	hipMemcpy(GPU_KSpaceIndexArray, KSpaceIndexArray, sizeof(int) * SizeZ, hipMemcpyHostToDevice);
	FrequencyAdjust << <dim3(SizeX, 1, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData, GPU_OCTFloatData, GPU_PXScale, GPU_KSpaceIndexArray, KSpaceOffset - CutValue, SizeX, 1, SizeZ);
	CheckCudaError();

	// 釋放記憶體
	hipFree(GPU_PXScale);
	hipFree(GPU_KSpaceIndexArray);
	hipFree(GPU_OCTRawData);
	delete[] KSpaceIndexArray;
	
	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "4. λ Space 轉成 K Space : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 5. cuFFT
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	hipfftHandle PlanHandle;
	hipfftComplex* GPU_ComplexData;

	// 這邊是創建 FFT 的 Handle & C2C 的 hipfftComplex
	int NX = SizeZ;
	int BatchSize = SizeX;
	hipfftPlan1d(&PlanHandle, NX, HIPFFT_C2C, BatchSize);
	hipMalloc(&GPU_ComplexData, sizeof(hipfftComplex) * NX * BatchSize);
	CheckCudaError();	

	// 把資料塞進 Complex Data 裡
	//gpuDataToComplex << <512, 4 >> > (GPU_OCTFloatData, GPU_ComplexData, NX * BatchSize, 0);
	DataToComplexData << <dim3(SizeX, 1, SizeZ / NumThreads), NumThreads >> > (GPU_OCTFloatData, GPU_ComplexData, OCTDataSize);
	CheckCudaError();

	// 執行 cuFFT(CUDA™ Fast Fourier Transform) 
	hipfftExecC2C(PlanHandle, GPU_ComplexData, GPU_ComplexData, HIPFFT_FORWARD);
	CheckCudaError();

	// 刪除鏡向(FFT轉完之後會兩邊對稱) & 搬移資料
	// 想知道更多：https://www.youtube.com/watch?v=spUNpyF58BY
	//gpuComplexToData << <512, 4 >> > (GPU_ComplexData, GPU_OCTFloatData, NX * BatchSize / 2, SizeZ, 0);
	ComplexDataToData << <dim3(SizeX, 1, SizeZ / NumThreads / 2), NumThreads >> > (GPU_ComplexData, GPU_OCTFloatData, SizeX, 1, SizeZ, OCTDataSize);
	CheckCudaError();

	// 刪除
	hipfftDestroy(PlanHandle);
	hipFree(GPU_ComplexData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "5. cuFFT: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 7. Normalize Data
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 要算出原始整條的強度值
	float *GPU_BrightnessArray;
	hipMalloc(&GPU_BrightnessArray, sizeof(float) * SizeX);
	ZCalcBrightness << <1, SizeX >> > (GPU_OCTFloatData, GPU_BrightnessArray, 1, SizeX, SizeZ / 2, StartIndex);
	CheckCudaError();

	// 算最大值
	float MaxValue = 0;
	float *GPU_MaxElement = thrust::max_element(thrust::device, GPU_OCTFloatData, GPU_OCTFloatData + OCTDataSize / 2);
	hipMemcpy(&MaxValue, GPU_MaxElement, sizeof(float), hipMemcpyDeviceToHost);
	CheckCudaError();

	// 最小值 (拿一塊不會使用的 GPU 部分，來做 Normalize) 
	// 拿一個正方形的區塊
	// TL－－－ｘ
	// ｜　　　｜
	// ｜　　　｜
	// ｘ－－－BR
	float MinValue = 0;
	for (int i = MinValuePixel_TL; i <= MinValuePixel_BR; i++)
	{
		// [first, last)
		int beginIndex = i * SizeZ / 2 + i;
		int endIndex = i * SizeZ / 2 + MinValuePixel_BR + 1;
		MinValue += thrust::reduce(thrust::device, GPU_OCTFloatData + beginIndex, GPU_OCTFloatData + endIndex);
	}
	MinValue /= (MinValuePixel_BR - MinValuePixel_TL + 1) * (MinValuePixel_BR - MinValuePixel_TL + 1);
	MinValue *= MinValueScalar;

	// 因為 Normaliza Data 要做一件事情是  除 (Max - Min) ，要預防他除以 0
	// 所以這邊先判斷兩個是不是位置一樣 (因為如果整個 array 值都一樣，Min & Max 給的位置都會一樣(以驗證過))
	assert(MaxValue != MinValue && "FFT後最大最小值一樣，資料有錯誤!!");
	NormalizeData << <dim3(SizeX, 1, SizeZ / NumThreads / 2), NumThreads >> > (GPU_OCTFloatData, MaxValue, MinValue, OCTDataSize / 2);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "7. Normalize Data: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 8. 轉成圖
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 圖片的資料
	uchar *GPU_UintDataArray;
	float* GPU_OCTSmoothData;
	hipMalloc(&GPU_UintDataArray, sizeof(uchar) * SizeX * 1 * SizeZ);
	hipMalloc(&GPU_OCTSmoothData, sizeof(float) * SizeX * 1 * SizeZ);
	CheckCudaError();

	// 轉圖片
	TransformToImageAndBorderData << <dim3(SizeX, 1, SizeZ / NumThreads / 2), NumThreads >> > (GPU_OCTFloatData, GPU_OCTSmoothData, GPU_UintDataArray, SizeX, 1, SizeZ / 2, SmoothSizeRange);
	CheckCudaError();

	// 設定一下其他參數
	size = 1;
	rows = SizeX;
	cols = SizeZ / 2;

	// 刪除記憶體
	hipFree(GPU_OCTFloatData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "8. 轉成圖: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 9. 邊界判斷
	// 目前邊界判斷沒有寫
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	#pragma region Init
	SaveDelete(PointType);
	PointType = new uchar[size * rows * cols];
	memset(PointType, 0, sizeof(uchar) * size * rows * cols);
	SaveDelete(PointType_1D);
	PointType_1D = new int[size * rows];
	memset(PointType_1D, 0, sizeof(int) * size * rows);

	// 點的型別
	uchar* GPU_PointType;
	hipMalloc(&GPU_PointType, sizeof(uchar) * size * rows * cols);
	hipMemset(GPU_PointType, 0, sizeof(uchar) * size * rows * cols);
	#pragma endregion
	#pragma region 抓取邊界
	assert(rows <= NumThreads && "rows 要小於 1024 的限制");

	// 找最大最小值 & 刪除過飽合的部分
	findMaxAndMinPeak << < size * rows * cols / NumThreads, NumThreads >> > (GPU_OCTSmoothData, GPU_BrightnessArray, GPU_PointType, size, rows, cols, MaxPeakThreshold, SatPeakThreshold);
	CheckCudaError();

	// Parse 一些連續最小值
	ParseMaxMinPeak << < size, rows >> > (GPU_PointType, size, rows, cols, StartIndex);
	CheckCudaError();

	// 抓出一維陣列
	int *GPU_PointType_BestN, *PointType_BestN;
	hipMalloc(&GPU_PointType_BestN, sizeof(int) * size * rows * ChooseBestN);
	//PickBestChoiceToArray << < size, rows >> > (GPU_OCTSmoothData, GPU_PointType, GPU_PointType_BestN, size, rows, cols, ChooseBestN, StartIndex, GoThroughThreshold);
	//CheckCudaError();

	// 連結點
	// 這個的大小 為 => 張數 * 250(rows) * 取幾個最大值(ChooseBestN個) * 每個最大值底下有 半徑個 (Raidus)  * 的下 N 排的幾個最大值(ChooseBestN) 
	int *GPU_Connect_Status;
	int ConnectStateSize = size * rows * ChooseBestN * ConnectRadius * ChooseBestN;
	hipMalloc(&GPU_Connect_Status, sizeof(int) * ConnectStateSize);
	hipMemset(GPU_Connect_Status, 0, sizeof(int) * ConnectStateSize);
	ConnectPointsStatus << < size * ChooseBestN , rows >> > (GPU_PointType_BestN, GPU_Connect_Status, size, rows, ChooseBestN, ConnectRadius);
	CheckCudaError();

	// 把資料傳回 CPU
	int *Connect_Status = new int[ConnectStateSize];
	PointType_BestN = new int[size * rows * ChooseBestN];
	hipMemcpy(PointType, GPU_PointType, sizeof(uchar) * size * rows * cols, hipMemcpyDeviceToHost);
	hipMemcpy(Connect_Status, GPU_Connect_Status, sizeof(int) * ConnectStateSize, hipMemcpyDeviceToHost);
	hipMemcpy(PointType_BestN, GPU_PointType_BestN, sizeof(int) * size * rows * ChooseBestN, hipMemcpyDeviceToHost);
	CheckCudaError();

	// 抓取最大的線
	GetSurface(PointType_BestN, Connect_Status);
	#pragma endregion

	// 刪除記憶體
	hipFree(GPU_PointType);
	hipFree(GPU_PointType_BestN);
	hipFree(GPU_Connect_Status);
	hipFree(GPU_OCTSmoothData);
	hipFree(GPU_BrightnessArray);

	delete[] Connect_Status;
	delete[] PointType_BestN;

	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "9. 抓取邊界: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 10. 抓下 GPU Data
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 刪除之前的資料
	SaveDelete(VolumeData);
	VolumeData = new uchar[SizeX * 1 * SizeZ];
	hipMemcpy(VolumeData, GPU_UintDataArray, sizeof(uchar) * SizeX * 1 * SizeZ / 2, hipMemcpyDeviceToHost);

	// 刪除 GPU
 	hipFree(GPU_UintDataArray);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "10. 抓下 GPU Data : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	
	// 結算
	#ifdef SHOW_TRCUDAV2_TOTAL_TIME
	totalTime = clock() - totalTime;
	cout << "轉換單張點雲: " << ((float)totalTime) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
}
void TRCudaV2::MultiRawDataToPointCloud(char* FileRawData, int DataSize, int SizeX, int SizeY, int SizeZ, long ShiftValue, double K_Step, int CutValue)
{
	// 計算時間
	#ifdef SHOW_TRCUDAV2_TOTAL_TIME
	totalTime = clock();
	#endif

	//////////////////////////////////////////////////////////////////////////
	// 步驟說明
	// 1. 上傳 GPU Data
	// 2. 一開始要把資料讀進來 (由於原本的資料都是 2個 Bytes 為一組，但 QT 目前是先用 GPU 轉換到 2個 Bytes)，和
	//    由於資料有 兩個 Channels，要相加除以2，可以去除雜訊 (由於原本的能量強度資料是使用三角波，所以會有去跟回兩個資料，就是把這兩筆資料相加除以 2)
	// 3. 用 5 次項去 Fit 一條曲線
	// 4. λ Space 轉成 K Space
	// 5. cuFFT
	// 6. 位移 Data
	// 6.5 要找出TopView  (這邊有多一個要找出TopView )
	// 7. 根據最大最小值來 Normalize 資料
	// 8. 轉成圖
	// 9. 邊界判斷
	// 10. 抓下 GPU Data
	//
	// 細節說明：
	// 1. 轉換 Function => X 快軸、Y 慢軸
	// 2. ShiftValue	=> TRIGGER DELAY位移(換FIBER，電線校正回來用的)
	// 3. K_Step		=> 深度(14.多mm對應 2.5的k step；可以考慮之後用2)(k step越大，z軸越深，但資料精細度越差；1~2.5)
	// 4. CutValue		=> OCT每個z軸，前面數據減去多少。原因是開頭的laser弱，干涉訊號不明顯，拿掉的資料會比較美。 (東元那邊的變數是 cuteValue XD)
	// 5. 只是這邊比上方的 Function 多了 SizeY 個
	// 6. 有多一個 找出TopView 
	//////////////////////////////////////////////////////////////////////////
	#pragma region 1. 上傳 GPU Data
	// 初始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	clock_t time = clock();
	#endif

	// GPU Data
	char* GPU_FileRawData;			// => 從檔案讀進來的 Raw Data
	int *GPU_OCTRawData_2Channel;	// => 這個是 OCT 掃完全部的 Raw Data (2Channels，如果"只用到一個" Channel 那就不會用到這個陣列)
	int *GPU_OCTRawData;			// => 這個是實際 Denoise 的 Data (也就是 CH1 + CH2 的資料) (如果"只有一個" Channel，就只會用到這個陣列)
	float *GPU_OCTFloatData;		// => 這個會用在兩個地方，一個是 K Space 的資料，一個是 FFT 後的資料

	// 是否是 2 Channels
	bool UseTwoChannels = (DataSize / SizeX / SizeY / SizeZ == 4);	// 2 Byte & 2 Channles

	// 原始資料
	hipMalloc(&GPU_FileRawData, sizeof(char) * DataSize);
	hipMemcpy(GPU_FileRawData, FileRawData, sizeof(char) * DataSize, hipMemcpyHostToDevice);
	CheckCudaError();

	// 判對是否使用 2 Chanels
	int OCTDataSize = SizeX * SizeY * SizeZ;
	if (UseTwoChannels)
		hipMalloc(&GPU_OCTRawData_2Channel, sizeof(int) * OCTDataSize * 2);
	hipMalloc(&GPU_OCTRawData, sizeof(int) * OCTDataSize);
	hipMalloc(&GPU_OCTFloatData, sizeof(float) * OCTDataSize);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "1. 上傳至 GPU: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion 
	#pragma region 2. 讀檔轉換
	//////////////////////////////////////////////////////////////////////////
	// 這邊的資料格式是這樣
	// ↗↘↗↘ 是一組 (↗代表掃描 0 ~ 250的一次資料)
	// 其中一個 ↗↘ 是一個三角波的資料
	// 但因為有兩個 channel 所以一組資料是 ↗↘↗↘
	//////////////////////////////////////////////////////////////////////////
	// 初始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 解出 2 Byte 的資料
	if (UseTwoChannels)
	{
		RawDataToOriginalData << < dim3(SizeX, SizeY, SizeZ / NumThreads * 2), NumThreads >> > (GPU_FileRawData, GPU_OCTRawData_2Channel, DataSize / 2);
		CheckCudaError();

		// 兩個 Channel 作 Denoise
		CombineTwoChannels_Multi << < dim3(SizeX, SizeY, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData_2Channel, GPU_OCTRawData, SizeX, SizeY, SizeZ);

		// 刪除
		hipFree(GPU_OCTRawData_2Channel);
	}
	else
		RawDataToOriginalData << < dim3(SizeX, SizeY, SizeZ / NumThreads), NumThreads >> > (GPU_FileRawData, GPU_OCTRawData, DataSize / 2);
	CheckCudaError();

	// 反掃的資料，Index 要反轉
	ReverseBackScanData << < dim3(SizeX / 2, SizeY / 2, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData, SizeX, SizeY, SizeZ);

	// 刪除 FileRaw Data
	hipFree(GPU_FileRawData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "2. 讀檔轉換: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 3. 用五次項去 Fitting
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 初始化 Matrix
	float* GPU_MatrixA;
	float* GPU_MatrixB;
	hipMalloc(&GPU_MatrixA, sizeof(float) * (NumPolynomial + 1) *(NumPolynomial + 1));
	hipMalloc(&GPU_MatrixB, sizeof(float) * (NumPolynomial + 1));

	// 先算平均
	int* FirstSizeZData = new int[SizeZ];
	hipMemcpy(FirstSizeZData, GPU_OCTRawData, sizeof(int) * SizeZ, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	float average = accumulate(FirstSizeZData, FirstSizeZData + SizeZ, 0.0) / SizeZ;
	delete[] FirstSizeZData;

	// 取得 Matrix
	GetMatrixA << <1, (NumPolynomial + 1) * (NumPolynomial + 1) >> > (GPU_OCTRawData, GPU_MatrixA, NumPolynomial, SizeZ);
	GetMatrixB << <1, NumPolynomial + 1 >> > (GPU_OCTRawData, GPU_MatrixB, average, NumPolynomial, SizeZ);
	CheckCudaError();

	float* MatrixA = new float[(NumPolynomial + 1) *(NumPolynomial + 1)];
	float* MatrixB = new float[(NumPolynomial + 1)];
	hipMemcpy(MatrixA, GPU_MatrixA, sizeof(float) * (NumPolynomial + 1) *(NumPolynomial + 1), hipMemcpyDeviceToHost);
	hipMemcpy(MatrixB, GPU_MatrixB, sizeof(float) * (NumPolynomial + 1), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// 解 Eigen 找 Fitting Function
	EigenUtility eigen;
	eigen.SetAverageValue(average);
	eigen.SolveByEigen(MatrixA, MatrixB, NumPolynomial);
	
	// 扣除那個 Function
	float* GPU_PolyValue;
	float* PolyValue = eigen.GetFunctionArray(SizeZ, average);
	hipMalloc(&GPU_PolyValue, sizeof(float) * SizeZ);
	hipMemcpy(GPU_PolyValue, PolyValue, sizeof(float) * SizeZ, hipMemcpyHostToDevice);
	MinusByFittingFunction << < dim3(SizeX, SizeY, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData, GPU_PolyValue, SizeZ);
	CheckCudaError();

	// 刪除多出來的
	hipFree(GPU_MatrixA);
	hipFree(GPU_MatrixB);
	hipFree(GPU_PolyValue);
	delete[] MatrixA;
	delete[] MatrixB;
	delete[] PolyValue;

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "3. 多項式去 Fitting : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 4. λ Space 轉成 K Space
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 初始化
	float* PX_Scale = new float[SizeZ];
	int* KSpaceIndexArray = new int[SizeZ];
	float* GPU_PXScale;
	int* GPU_KSpaceIndexArray;
	hipMalloc(&GPU_PXScale,			sizeof(float) * SizeZ);
	hipMalloc(&GPU_KSpaceIndexArray,	sizeof(int) * SizeZ);

	// 設定一些系數
	int OffsetBegin = 800;

	// 算出 PXScale 的 Array
	ComputePXScale << <SizeZ / NumThreads, NumThreads >> > (GPU_PXScale, OffsetBegin, ShiftValue, K_Step, SizeZ);
	CheckCudaError();

	// 抓下來準備算 K Space Index (由於這邊如果使用 GPU 去做，會導致大部分的 thread 在等最大工作量的 thread，所以這裡 CPU 做會比較快)
	hipMemcpy(PX_Scale, GPU_PXScale, sizeof(float) * SizeZ, hipMemcpyDeviceToHost);

	// 算 K Space 的對應 Array
	int index = 1;
	int KSpaceOffset = PX_Scale[SizeZ - 1];
	for (int i = 0; i <= KSpaceOffset; i++)
	{
		while (i >= PX_Scale[index])
		{
			index++;
		}
		KSpaceIndexArray[i] = index;
	}
	for (int i = KSpaceOffset + 1; i < SizeZ; i++)
		KSpaceIndexArray[i] = -1;

	// 由於 K Space 不是線性關係，所以要從 KSpaceIndexArray，找 Index，再從左右兩個點中，內插出實際在這個 Index 的值
	hipMemcpy(GPU_KSpaceIndexArray, KSpaceIndexArray, sizeof(int) * SizeZ, hipMemcpyHostToDevice);
	FrequencyAdjust << <dim3(SizeX, SizeY, SizeZ / NumThreads), NumThreads >> > (GPU_OCTRawData, GPU_OCTFloatData, GPU_PXScale, GPU_KSpaceIndexArray, KSpaceOffset - CutValue, SizeX, SizeY, SizeZ);
	CheckCudaError();

	// 釋放記憶體
	hipFree(GPU_PXScale);
	hipFree(GPU_KSpaceIndexArray);
	hipFree(GPU_OCTRawData);
	delete[] KSpaceIndexArray;

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "4. λ Space 轉成 K Space : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 5. cuFFT
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	hipfftHandle PlanHandle;
	hipfftComplex* GPU_ComplexData;

	// 這邊是創建 FFT 的 Handle & C2C 的 hipfftComplex
	int NX = SizeZ;
	int BatchSize = SizeX * SizeY;
	hipfftPlan1d(&PlanHandle, NX, HIPFFT_C2C, BatchSize);
	hipMalloc(&GPU_ComplexData, sizeof(hipfftComplex) * NX * BatchSize);
	CheckCudaError();	

	// 把資料塞進 Complex Data 裡
	DataToComplexData << <dim3(SizeX, SizeY, SizeZ / NumThreads), NumThreads >> > (GPU_OCTFloatData, GPU_ComplexData, OCTDataSize);
	CheckCudaError();

	// 執行 cuFFT(CUDA™ Fast Fourier Transform) 
	hipfftExecC2C(PlanHandle, GPU_ComplexData, GPU_ComplexData, HIPFFT_FORWARD);
	CheckCudaError();

	// 刪除鏡向(FFT轉完之後會兩邊對稱) & 搬移資料
	// 想知道更多：https://www.youtube.com/watch?v=spUNpyF58BY
	ComplexDataToData << <dim3(SizeX, SizeY, SizeZ / NumThreads / 2), NumThreads >> > (GPU_ComplexData, GPU_OCTFloatData, SizeX, SizeY, SizeZ, OCTDataSize);
	CheckCudaError();

	// 刪除
	hipfftDestroy(PlanHandle);
	hipFree(GPU_ComplexData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "5. cuFFT: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 6. 位移 Data
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	float* GPU_ShiftData;
	hipMalloc(&GPU_ShiftData, sizeof(float) * OCTDataSize / 2);		// 因為一半相同，所以去掉了
	
	// 這邊也是
	ShiftFinalData << <dim3(SizeX, SizeY, SizeZ / NumThreads / 2), NumThreads >> > (GPU_OCTFloatData, GPU_ShiftData, SizeX, SizeY, SizeZ / 2, OCTDataSize / 2);
	CheckCudaError();

	// 刪除記憶體
	hipFree(GPU_OCTFloatData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "6. 搬移資料: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 6.5 TopView 
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 這邊會抓出TopView 
	float* GPU_OtherSideData;
	hipMalloc(&GPU_OtherSideData, sizeof(float) * OCTDataSize / 2);
	GetOtherSideView << <SizeX, SizeY >> > (GPU_ShiftData, GPU_OtherSideData, SizeX, SizeY, SizeZ / 2);
	CheckCudaError();
	hipDeviceSynchronize();

	// 找最大值
	float MaxValue = 0, MinValue = 0;
	float *GPU_MaxElement = thrust::max_element(thrust::device, GPU_OtherSideData, GPU_OtherSideData + SizeX * SizeY);
	float *GPU_MinElement = thrust::min_element(thrust::device, GPU_OtherSideData, GPU_OtherSideData + SizeX * SizeY);
	hipMemcpy(&MaxValue, GPU_MaxElement, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&MinValue, GPU_MinElement, sizeof(float), hipMemcpyDeviceToHost);

	NormalizeData << < SizeX, SizeY >> > (GPU_OtherSideData, MaxValue, MinValue, SizeX * SizeY);
	CheckCudaError();

	// 將 Top View 的圖，部會因為亮度而受影響
	float MeanValue = thrust::reduce(thrust::device, GPU_OtherSideData, GPU_OtherSideData + SizeX * SizeY) / SizeX / SizeY;

	uchar* GPU_UintOtherSideData;
	hipMalloc(&GPU_UintOtherSideData, sizeof(uchar) * SizeX * SizeY);
	TransformOtherSideDataToImage << <SizeX, SizeY >> > (GPU_OtherSideData, GPU_UintOtherSideData, MeanValue, OtherSideMean, SizeX, SizeY);
	CheckCudaError();

	// 刪除記憶體
	hipFree(GPU_OtherSideData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "6.5. TopView 產生: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 7. Normalize Data
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	float* CPU_ShiftData;
	hipMalloc(&CPU_ShiftData, sizeof(float) * OCTDataSize / 2);		// 因為一半相同，所以去掉了
	NonNormalizeImage = new float[sizeof(float) * OCTDataSize / 2];

	GpuDataToCpuData << <dim3(SizeX, SizeY, SizeZ / NumThreads / 2), NumThreads >> > (GPU_ShiftData, CPU_ShiftData, OCTDataSize / 2);

	hipMemcpy(NonNormalizeImage, CPU_ShiftData, sizeof(float) * OCTDataSize / 2, hipMemcpyDeviceToHost);
	hipFree(CPU_ShiftData);

	//TransfromNonNormalizeData(NonNormalizeImage);

	//for (int i = 0; i < 1024; i++)cout << i << " : " << NonNormalizeImage[i] << endl;

	// 算最大值
	MaxValue = 0;
	GPU_MaxElement = thrust::max_element(thrust::device, GPU_ShiftData, GPU_ShiftData + OCTDataSize / 2);
	hipMemcpy(&MaxValue, GPU_MaxElement, sizeof(float), hipMemcpyDeviceToHost);
	CheckCudaError();
	hipDeviceSynchronize();

	// 最小值 (拿一塊不會使用的 GPU 部分，來做 Normalize)
	// 拿一個正方形的區塊
	// TL－－－ｘ
	// ｜　　　｜
	// ｜　　　｜
	// ｘ－－－BR
	MinValue = 0;
	for (int i = MinValuePixel_TL; i <= MinValuePixel_BR; i++)
	{
		// [first, last)
		int beginIndex = SizeX * SizeZ / 2 + i * SizeZ / 2 + i;
		int endIndex = SizeX * SizeZ / 2 + i * SizeZ / 2 + MinValuePixel_BR + 1;
		MinValue += thrust::reduce(thrust::device, GPU_ShiftData + beginIndex, GPU_ShiftData + endIndex);
	}
	MinValue /= (MinValuePixel_BR - MinValuePixel_TL + 1) * (MinValuePixel_BR - MinValuePixel_TL + 1);
	MinValue *= MinValueScalar;

	// 因為 Normaliza Data 要做一件事情是  除 (Max - Min) ，要預防他除以 0
	// 所以這邊先判斷兩個是不是位置一樣 (因為如果整個 array 值都一樣，Min & Max 給的位置都會一樣(以驗證過))
	assert(MaxValue != MinValue && "FFT後最大最小值一樣，資料有錯誤!!");
	NormalizeData << <dim3(SizeX, SizeY, SizeZ / NumThreads / 2), NumThreads >> > (GPU_ShiftData, MaxValue, MinValue, OCTDataSize / 2);
	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "7. Normalize Data: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 8. 轉成圖
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	// 圖片的資料
	uchar *GPU_UintDataArray;
	float* GPU_OCTSmoothData;
	hipMalloc(&GPU_UintDataArray, sizeof(uchar) * SizeX * SizeY * SizeZ);
	hipMalloc(&GPU_OCTSmoothData, sizeof(float) * SizeX * SizeY * SizeZ);
	CheckCudaError();

	// 轉圖片
	TransformToImageAndBorderData << <dim3(SizeX, SizeY, SizeZ / NumThreads / 2), NumThreads >> > (GPU_ShiftData, GPU_OCTSmoothData, GPU_UintDataArray, SizeX, SizeY, SizeZ / 2, SmoothSizeRange);
	CheckCudaError();

	// 設定一下其他參數
	size = SizeY;
	rows = SizeX;
	cols = SizeZ / 2;

	// 刪除記憶體
	hipFree(GPU_ShiftData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "8. 轉成圖: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 9. 抓取邊界
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif

	#pragma region Init
	SaveDelete(PointType);
	PointType = new uchar[size * rows * cols];
	memset(PointType, 0, sizeof(uchar) * size * rows * cols);
	SaveDelete(PointType_1D);
	PointType_1D = new int[size * rows];
	memset(PointType_1D, 0, sizeof(int) * size * rows);

	// 點的型別
	uchar* GPU_PointType;
	hipMalloc(&GPU_PointType, sizeof(uchar) * size * rows * cols);
	hipMemset(GPU_PointType, 0, sizeof(uchar) * size * rows * cols);
	#pragma endregion
	#pragma region 抓取邊界
	assert(rows <= NumThreads && "rows 要小於 1024 的限制");

	// 要算出原始整條的強度值
	float *GPU_BrightnessArray;
	hipMalloc(&GPU_BrightnessArray, sizeof(float) * size * rows);
	ZCalcBrightness << <size, rows >> > (GPU_OCTSmoothData, GPU_BrightnessArray, size, rows, cols, StartIndex);
	CheckCudaError();
	
	// 找最大最小值 & 刪除過飽合的部分
	findMaxAndMinPeak << < size * rows * cols / NumThreads, NumThreads >> > (GPU_OCTSmoothData, GPU_BrightnessArray, GPU_PointType, size, rows, cols, MaxPeakThreshold, SatPeakThreshold);
	CheckCudaError();

	// Parse 一些連續最小值
	ParseMaxMinPeak << < size, rows >> > (GPU_PointType, size, rows, cols, StartIndex);
	CheckCudaError();

	// 抓出一維陣列
	int *GPU_PointType_BestN, *PointType_BestN;
	float* GPU_CandidateGap;
	hipMalloc(&GPU_PointType_BestN, sizeof(int) * size * rows * ChooseBestN);
	hipMalloc(&GPU_CandidateGap, sizeof(float) * size * rows * cols);				// 暫時用來存 Gap 的記憶體
	hipMemset(GPU_CandidateGap, 0, sizeof(float) * size * rows * cols);
	hipMemset(GPU_PointType_BestN, 0, sizeof(int) * size * rows * ChooseBestN);
	PickBestChoiceToArray << < size, rows >> > (GPU_OCTSmoothData, GPU_PointType, GPU_CandidateGap, GPU_PointType_BestN, size, rows, cols, ChooseBestN, StartIndex, GoThroughThreshold);
	CheckCudaError();

	// 算出 Neighbor 數目的陣列
	float* GPU_NeighborCountArray;
	hipMalloc(&GPU_NeighborCountArray, sizeof(float) * size * rows * ChooseBestN);
	CalcNeighbor << <size, rows >> > (GPU_PointType_BestN, GPU_NeighborCountArray, size, rows, cols, ChooseBestN, DenoiseWindowsRadius);
	CheckCudaError();

	// 連結點
	// 這個的大小 為 => 張數 * 250(rows) * 取幾個最大值(ChooseBestN個) * 每個最大值底下有 半徑個 (Raidus)  * 的下 N 排的幾個最大值(ChooseBestN) 
	int *GPU_Connect_Status;
	int ConnectStateSize = size * rows * ChooseBestN * ConnectRadius * ChooseBestN;
	hipMalloc(&GPU_Connect_Status, sizeof(int) * ConnectStateSize);
	hipMemset(GPU_Connect_Status, 0, sizeof(int) * ConnectStateSize);
	ConnectPointsStatus << < size * ChooseBestN, rows >> > (GPU_PointType_BestN, GPU_Connect_Status, size, rows, ChooseBestN, ConnectRadius);
	CheckCudaError();

	// 把資料傳回 CPU
	int *Connect_Status = new int[ConnectStateSize];
	PointType_BestN = new int[size * rows * ChooseBestN];
	hipMemcpy(PointType, GPU_PointType, sizeof(uchar) * size * rows * cols, hipMemcpyDeviceToHost);
	hipMemcpy(Connect_Status, GPU_Connect_Status, sizeof(int) * ConnectStateSize, hipMemcpyDeviceToHost);
	hipMemcpy(PointType_BestN, GPU_PointType_BestN, sizeof(int) * size * rows * ChooseBestN, hipMemcpyDeviceToHost);
	CheckCudaError();

	// 抓取最大的線
	GetSurface(PointType_BestN, Connect_Status);
	#pragma endregion

	// 刪除記憶體
	hipFree(GPU_PointType);
	hipFree(GPU_PointType_BestN);
	hipFree(GPU_Connect_Status);
	hipFree(GPU_OCTSmoothData);
	hipFree(GPU_BrightnessArray);
	hipFree(GPU_CandidateGap);
	hipFree(GPU_NeighborCountArray);

	delete[] Connect_Status;
	delete[] PointType_BestN;

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "9. 抓取邊界: " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion
	#pragma region 10. 抓下 GPU Data
	// 開始
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock();
	#endif
	
	// 刪除之前的資料
	SaveDelete(VolumeData);
	VolumeData = new uchar[SizeX * SizeY * SizeZ / 2];
	hipMemcpy(VolumeData, GPU_UintDataArray, sizeof(uchar) * SizeX * SizeY * SizeZ / 2, hipMemcpyDeviceToHost);

	SaveDelete(VolumeData_OtherSide);
	VolumeData_OtherSide = new uchar[SizeX * SizeY];
	hipMemcpy(VolumeData_OtherSide, GPU_UintOtherSideData, sizeof(uchar) * SizeX * SizeY, hipMemcpyDeviceToHost);

	// 刪除 GPU
	hipFree(GPU_UintDataArray);
	hipFree(GPU_UintOtherSideData);

	// 結算
	#ifdef SHOW_TRCUDAV2_DETAIL_TIME
	time = clock() - time;
	cout << "10. 抓下 GPU Data : " << ((float)time) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
	#pragma endregion

	// 結算
	#ifdef SHOW_TRCUDAV2_TOTAL_TIME
	totalTime = clock() - totalTime;
	cout << "轉換多張點雲: " << ((float)totalTime) / CLOCKS_PER_SEC << " sec" << endl;
	#endif
}

// 拿出圖片
vector<Mat> TRCudaV2::TransfromMatArray(bool SaveBorder = false)
{
	// 轉換到 Mat
	vector<Mat> ImgArray;
	for (int i = 0; i < size; i++)
	{
		// 根據 Offset 拿圖片
		Mat img(rows, cols, CV_8U, VolumeData + i * rows * cols);
		cvtColor(img, img, CV_GRAY2BGR);

		// 丟進堆疊
		ImgArray.push_back(img);
	}
	if (SaveBorder)
	{
		// Debug 所有的 peak
		/*for (int i = 0; i < size; i++)
		for (int j = 0; j < rows * cols; j++)
		{
		int offsetIndex = i * rows * cols;
		int rowIndex = j / cols;
		int colIndex = j % cols;

		Vec3b color(0, 0, 0);
		if (PointType[offsetIndex + j] == 1)
		color = Vec3b(0, 255, 255);
		else if (PointType[offsetIndex + j] == 2)
		color = Vec3b(255, 255, 255);
		ImgArray[i].at<Vec3b>(rowIndex, colIndex) = color;
		}*/

		// 只抓出最後的邊界
		for (int i = 0; i < size; i++)
			for (int j = 0; j < rows; j++)
			{
				int index = i * rows + j;
				if (PointType_1D[index] != -1)
				{
					Point contourPoint(PointType_1D[index], j);
					circle(ImgArray[i], contourPoint, 2, Scalar(0, 255, 255), CV_FILLED);
				}
			}
	}
	return ImgArray;
}
Mat TRCudaV2::TransformToOtherSideView()
{
	assert(size > 1 && "這段一定要大於一張圖");

	Mat img(rows, size, CV_8U, VolumeData_OtherSide);
	cvtColor(img, img, CV_GRAY2BGR);
	return img;
}
void TRCudaV2::CopySingleBorder(int* LastArray)
{
	assert(LastArray != NULL && PointType_1D != NULL && size == 1 && "要先初始化 Array 和要做轉點雲的部分!!");	// assert 抓出 call 錯的可能性 (這邊只能單張)
	memcpy(LastArray, PointType_1D, sizeof (int) * size * rows);
}
void TRCudaV2::CopyBorder(int* BorderArray)
{
	assert(BorderArray != NULL && PointType_1D != NULL && size != 1 && "要先初始化 Array 和要做轉點雲的部分!!");	// assert 抓出 call 錯的可能性 (這邊要多張)
	memcpy(BorderArray, PointType_1D, sizeof(int) * size * rows);
}
bool TRCudaV2::ShakeDetect_Single(int* LastArray, bool ShowDebugMessage)
{
	// 設定變數
	int voteNum = 0;								// 有效票數
	float MoveDis = 0;								// 移動的總共距離

	// 跑每一個點
	for (int i = 0; i < rows; i++)
	{
		if (PointType_1D[i] != -1 && LastArray[i] != -1)
		{
			MoveDis += abs(PointType_1D[i] - LastArray[i]);
			voteNum++;
		}
	}

	// 判斷是否有有效資料
	if (voteNum > OCT_Valid_VoteNum)
	{
		MoveDis /= voteNum;

		// Debug Message
		if(ShowDebugMessage)
			cout << "晃動距離(pixel): " << (float)MoveDis << endl;

		// 這邊是代表沒有晃動
		if (MoveDis < OCT_Move_Threshold)
			return false;
	}
	return true;
}
bool TRCudaV2::ShakeDetect_Multi(bool UsePreiseThreshold, bool ShowDebugMessage)
{
	// 找 60 ~ 200 裡面有效的有沒有斷層
	int voteNum = 0;								// 有效票數
	float MoveDis = 0;								// 移動的總共距離
	
	// Reverse 後的 0 ~ 250
	for (int i = 60; i < 200; i++)
	{
		bool IsMove = false;

		// 這邊先預設給這個值，後面會換掉
		int leftIndex = 124 * rows + i;				// 第 124 張
		int rightIndex = 125 * rows + i;			// 第 125 張

		// 從中間往前找
		for (int j = size / 2 - 1; j >= 0; j--)
			if (PointType_1D[j * rows + i] != -1)
			{
				leftIndex = j * rows + i;
				break;
			}
		
		// 從中間像後找
		for (int j = size / 2; j < size; j++)
			if (PointType_1D[j] != -1)
			{
				rightIndex = j * rows + i;
				break;
			}

		int leftY = PointType_1D[leftIndex];
		int rightY = PointType_1D[rightIndex];

		// 確認有效票數
		if (PointType_1D[leftIndex] != -1 &&
			PointType_1D[rightIndex] != -1)
		{
			int DisMid = abs(PointType_1D[rightIndex] - PointType_1D[leftIndex]);
			MoveDis += DisMid;
			voteNum++;
		}

	}

	// 判斷是否有有效資料
	if (voteNum > OCT_Valid_VoteNum)
	{
		MoveDis /= voteNum;

		// Debug Message
		if (ShowDebugMessage)
			cout << "晃動距離(pixel): " << (float)MoveDis << endl;

		// 這邊是代表沒有晃動
		if (UsePreiseThreshold)
		{
			// 用較輕確的結果
			if (MoveDis < OCT_Move_Precise_Threshold)
				return false;
		}
		else
		{
			// 用較不精確的結果
			if (MoveDis < OCT_Move_Threshold)
				return false;
		}
	}
	else if (ShowDebugMessage)
		cout << "資料量太少!!" << endl;
	return true;
}
float* TRCudaV2::TransfromNonNormalizeData() {
	return NonNormalizeImage;
}

//////////////////////////////////////////////////////////////////////////
// Helper Function
//////////////////////////////////////////////////////////////////////////
void TRCudaV2::GetSurface(int *PointType_BestN, int *Connect_Status)
{
	// 選 N 個
	#pragma omp parallel for //num_thread(4)
	for (int i = 0; i < size; i++)
	{
		// 每個 10 段下去 Sample
		int RowGap = rows / 10;
		vector<vector<ConnectInfo>> StatusVector;
		
		for (int j = 0; j < rows; j += RowGap)
			for (int chooseNIndex = 0; chooseNIndex < ChooseBestN; chooseNIndex++)
			{
				int begin = j;
				int end = j;

				// 代表這個點沒有東西，所以略過
				if (PointType_BestN[i * rows * ChooseBestN + j * ChooseBestN + chooseNIndex] == -1)
					continue;

				// 連接狀況
				vector<ConnectInfo> Connect;

				#pragma region 往上找
				// 先加上自己
				ConnectInfo info;
				info.rowIndex = j;
				info.chooseIndex = chooseNIndex;
				Connect.push_back(info);

				int FindIndex = j;
				int FindChooseIndex = chooseNIndex;
				bool IsFind = true;
				while (IsFind && FindIndex > 0)
				{
					int minMoveIndex = -1;
					int minChooseIndex = -1;
					int tempValue = ConnectRadius * ConnectRadius;
					for (int k = 1; k < ConnectRadius; k++)
						for (int nextChooseNIndex = 0; nextChooseNIndex < ChooseBestN; nextChooseNIndex++)
						{
							int index = i * rows * ChooseBestN * ConnectRadius * ChooseBestN +					// Size
										(FindIndex - k) * ChooseBestN * ConnectRadius * ChooseBestN +			// Rows
										nextChooseNIndex * ConnectRadius * ChooseBestN +						// 現在在的 Top N 的點 (這邊要注意，這邊應該要放的是 要找的那個點的 ChooseIndex)
										k * ChooseBestN +														// 半徑
										FindChooseIndex;
							if (FindIndex - k >= 0 && Connect_Status[index] != 0 && tempValue > Connect_Status[index])
							{
								tempValue = Connect_Status[index];
								minMoveIndex = k;
								minChooseIndex = nextChooseNIndex;
							}
						}

					// 判斷是否有找到，找到就繼續找
					if (minMoveIndex != -1)
					{
						// 更便位置
						FindIndex = FindIndex - minMoveIndex;
						FindChooseIndex = minChooseIndex;

						// 丟進陣列
						info.rowIndex = FindIndex;
						info.chooseIndex = minChooseIndex;
						Connect.push_back(info);

						// 有找到
						IsFind = true;
					}
					else
						IsFind = false;
				}
				#pragma endregion
				#pragma region 往下找
				FindIndex = j;
				FindChooseIndex = chooseNIndex;
				while (IsFind && FindIndex < rows - 1)
				{
					int minMoveIndex = -1;
					int minChooseIndex = -1;				
					int tempValue = ConnectRadius * ConnectRadius;
					for (int k = 1; k < ConnectRadius; k++)
						for (int nextChooseNIndex = 0; nextChooseNIndex < ChooseBestN; nextChooseNIndex++)
						{
							int index = i * rows * ChooseBestN * ConnectRadius * ChooseBestN +					// Size
										FindIndex * ChooseBestN * ConnectRadius * ChooseBestN +					// Rows
										FindChooseIndex * ConnectRadius * ChooseBestN +							// 現在在的 Top N 的點
										k * ChooseBestN +														// 半徑
										nextChooseNIndex;
							if (FindIndex + k < rows && Connect_Status[index] != 0 && tempValue > Connect_Status[index])
							{
								tempValue = Connect_Status[index];
								minMoveIndex = k;
								minChooseIndex = nextChooseNIndex;
							}
						}

					// 判斷是否有找到，找到就繼續找
					if (minMoveIndex != -1)
					{
						// 更便位置
						FindIndex = FindIndex + minMoveIndex;
						FindChooseIndex = minChooseIndex;

						// 丟進陣列
						info.rowIndex = FindIndex;
						info.chooseIndex = minChooseIndex;
						Connect.push_back(info);

						// 有找到
						IsFind = true;
					}
					else
						IsFind = false;
				}
				#pragma endregion
			
				// 判斷是否有連出東西，如果連出來的東西大於 1
				if (Connect.size() > 1)
				{
					// 由小排到大
					sort(Connect.begin(), Connect.end(), SortByRows);
					StatusVector.push_back(Connect);
				}
			}
		

		// 前面的幾個張數，可能會找不到點，所以跳過處理
		if (StatusVector.size() == 0)
		{
			memset(&PointType_1D[i * rows], -1, sizeof(int) * rows);
			continue;
		}

		// 排序之後取最大
		sort(StatusVector.begin(), StatusVector.end(), SortByVectorSize);


		// 超出不重疊的最好連接方法 (最多取前三個)
		vector<int> BestCandidate;
		int Begin = rows;
		int End = 0;
		for (int j = 0; j < StatusVector.size() && j < 3; j++)
		{
			int CurrentBegin = StatusVector[j][0].rowIndex;
			int CurrentEnd = StatusVector[j][StatusVector[j].size() - 1].rowIndex;

			if (Begin > CurrentBegin)
			{
				Begin = min(Begin, CurrentBegin);
				End = max(End, CurrentEnd);
				BestCandidate.push_back(j);
			}

			if (End < CurrentEnd)
			{
				Begin = min(Begin, CurrentBegin);
				End = max(End, CurrentEnd);
				BestCandidate.push_back(j);
			}
		}

		// 加到裡面
		for (int j = 1; j < BestCandidate.size(); j++)
			if (StatusVector[BestCandidate[j]].size() >= 3)
				for (int k = 0; k < StatusVector[BestCandidate[j]].size(); k++)
					StatusVector[0].push_back(StatusVector[j][k]);

		vector<ConnectInfo> LineVector = StatusVector[0];
		int index = 0;				// LineVector Index
		for (int j = 0; j < rows; j++)
		{
			int Type1D_Index = i * rows + j;
			if (LineVector[index].rowIndex != j)
				PointType_1D[Type1D_Index] = -1;
			else if (LineVector[index].rowIndex == j)
			{
				int BestN_Index = i * rows * ChooseBestN +							// 張
								LineVector[index].rowIndex * ChooseBestN +			// row
								LineVector[index].chooseIndex;						// ChooseIndex

				// 放進 PointType
				PointType_1D[j + i * rows] = PointType_BestN[BestN_Index];
				index++;

				if (index >= LineVector.size())
				{
					for (int k = j + 1; k < rows; k++)
						PointType_1D[k + i * rows] = -1;
					break;
				}
			}
		}
	}

	// Smooth
	int* tempPointType_1D = new int[size * rows];
	for (int i = 0; i < size; i++)
		for (int j = 0; j < rows; j ++)
		{
			int totalPoint = 0;
			int totalZ = 0;
			int index = i * rows + j;
			if (PointType_1D[index] == -1)
			{
				tempPointType_1D[index] = -1;
				continue;
			}

			for (int k = -DenoiseWindowsRadius; k <= DenoiseWindowsRadius; k++)
				for (int l = -DenoiseWindowsRadius; l <= DenoiseWindowsRadius; l++)
				{
					int currentI = i + k;
					int currentJ = j + l;
					if (0 <= currentI && currentI < size && 
						0 <= currentJ && currentJ < rows)
					{
						int currentIndex = currentI *rows + currentJ;
						if (PointType_1D[currentIndex] != -1)
						{
							totalPoint++;
							totalZ += PointType_1D[currentIndex];
						}
					}
				}
			tempPointType_1D[index] = totalZ / totalPoint;
		}

	memcpy(PointType_1D, tempPointType_1D, sizeof(int) * size * rows);
	delete[] tempPointType_1D;
}
bool TRCudaV2::SortByRows(ConnectInfo left, ConnectInfo right)
{
	return left.rowIndex < right.rowIndex;
}
bool TRCudaV2::SortByVectorSize(vector<ConnectInfo> left, vector<ConnectInfo> right)
{
	return right.size() < left.size();
}
void TRCudaV2::CheckCudaError()
{
	hipError_t GPU_Error = hipGetLastError();
	if (GPU_Error != hipSuccess)
	{
		cout << hipGetErrorString(GPU_Error) << endl;
		assert(false);
		exit(-1);
	}
}
void TRCudaV2::SaveDelete(void* pointer)
{
	if (pointer != NULL)
		delete[] pointer;
}